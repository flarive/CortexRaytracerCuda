#include "hip/hip_runtime.h"
#include <iostream>

// cuda
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include <thrust/random.h>


#include "misc/vector3.cuh"
#include "misc/bvh_node.cuh"


#include "primitives/hittable_list.cuh"

#include "textures/texture.cuh"
#include "textures/solid_color_texture.cuh"
#include "textures/gradient_texture.cuh"
#include "textures/checker_texture.cuh"
#include "textures/image_texture.cuh"
#include "textures/bump_texture.cuh"
#include "textures/normal_texture.cuh"
#include "textures/alpha_texture.cuh"
#include "textures/emissive_texture.cuh"
#include "textures/perlin_noise_texture.cuh"



#include "materials/diffuse_light.cuh"
#include "materials/diffuse_spot_light.cuh"
#include "materials/lambertian.cuh"
#include "materials/metal.cuh"
#include "materials/dielectric.cuh"
#include "materials/isotropic.cuh"
#include "materials/anisotropic.cuh"
#include "materials/oren_nayar.cuh"
#include "materials/phong.cuh"



#include "primitives/aarect.cuh"
#include "primitives/box.cuh"
#include "primitives/sphere.cuh"
#include "primitives/quad.cuh"
#include "primitives/volume.cuh"
#include "primitives/torus.cuh"
#include "primitives/cylinder.cuh"
#include "primitives/cone.cuh"
#include "primitives/disk.cuh"
#include "primitives/triangle.cuh"

#include "primitives/translate.cuh"
#include "primitives/rotate.cuh"
#include "primitives/scale.cuh"
#include "primitives/flip_normals.cuh"


#include "lights/light.cuh"
#include "lights/omni_light.cuh"
#include "lights/directional_light.cuh"
#include "lights/spot_light.cuh"

#include "cameras/camera.cuh"
#include "cameras/perspective_camera.cuh"
#include "cameras/orthographic_camera.cuh"

#include "samplers/sampler.cuh"
#include "samplers/random_sampler.cuh"

#include "utilities/bitmap_image.cuh"


#include "scenes/scene_config.h"

#include "scene_factory.cuh"



#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb/stb_image.h>
#include <stb/stb_image_write.h>




// https://github.com/Belval/raytracing

bool isGpuAvailable(hipDeviceProp_t& prop)
{
    int devicesCount;
    hipGetDeviceCount(&devicesCount);
    for (int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
    {
        hipGetDeviceProperties(&prop, deviceIndex);
        if (prop.major >= 2 && prop.minor >= 0)
        {
            printf("[INFO] Use GPU device %d %s\n", deviceIndex, prop.name);
            printf("[INFO] Number of multiprocessors on device: %d\n", prop.multiProcessorCount);
            printf("[INFO] Total global memory (Gbytes) %.1f\n", (float)(prop.totalGlobalMem) / 1024.0 / 1024.0 / 1024.0);
            printf("[INFO] Max grid size: %i x %i x %i\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
            printf("[INFO] Max block size: %i x %i x %i\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
            printf("[INFO] Max number of threads per block: %i\n", prop.maxThreadsPerBlock);

            hipSetDevice(deviceIndex);

            return true;
        }
    }

    std::cout << "[ERROR] No Nvidia Cuda GPU device found" << std::endl;
    return false;
}


#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' " << hipGetErrorString(result) << "\n";
        // reset device before terminating
        hipDeviceReset();
        exit(99);
    }
}

// move in cuda helpers
__device__ int strcmp_device(const char* str1, const char* str2)
{
    while (*str1 && (*str1 == *str2)) {
        str1++;
        str2++;
    }
    return *(unsigned char*)str1 - *(unsigned char*)str2;
}

__device__ texture* fetchTexture(sceneConfig* sceneCfg, bitmap_image** images, const char* textureName)
{
    for (int i = 0; i < sceneCfg->texturesCfg.solidColorTextureCount; i++)
    {
        solidColorTextureConfig solidColorTexture = sceneCfg->texturesCfg.solidColorTextures[i];
        printf("[GPU] solidColorTexture%d %s %g/%g/%g\n", i,
            solidColorTexture.name,
            solidColorTexture.rgb.r(), solidColorTexture.rgb.g(), solidColorTexture.rgb.b());

        if (strcmp_device(solidColorTexture.name, textureName) == 0)
        {
            return new solid_color_texture(solidColorTexture.rgb);
        }
    }

    for (int i = 0; i < sceneCfg->texturesCfg.gradientColorTextureCount; i++)
    {
        gradientColorTextureConfig gradientColorTexture = sceneCfg->texturesCfg.gradientColorTextures[i];
        printf("[GPU] gradientColorTexture%d %s %g/%g/%g %g/%g/%g %d %d\n", i,
            gradientColorTexture.name,
            gradientColorTexture.color1.r(), gradientColorTexture.color1.g(), gradientColorTexture.color1.b(),
            gradientColorTexture.color2.r(), gradientColorTexture.color2.g(), gradientColorTexture.color2.b(),
            gradientColorTexture.vertical,
            gradientColorTexture.hsv);

        if (strcmp_device(gradientColorTexture.name, textureName) == 0)
        {
            return new gradient_texture(gradientColorTexture.color1, gradientColorTexture.color2, gradientColorTexture.vertical, gradientColorTexture.hsv);
        }
    }

    for (int i = 0; i < sceneCfg->texturesCfg.imageTextureCount; i++)
    {
        imageTextureConfig imageTexture = sceneCfg->texturesCfg.imageTextures[i];
        printf("[GPU] imageTexture%d %s %s\n", i,
            imageTexture.name,
            imageTexture.filepath);

        if (strcmp_device(imageTexture.name, textureName) == 0)
        {
            bitmap_image img = *(images[0]);
            return new image_texture(img);
        }
    }

    for (int i = 0; i < sceneCfg->texturesCfg.checkerTextureCount; i++)
    {
        checkerTextureConfig checkerTexture = sceneCfg->texturesCfg.checkerTextures[i];
        printf("[GPU] checkerTexture%d %s %g/%g/%g %g/%g/%g %s %s %g\n", i,
            checkerTexture.name,
            checkerTexture.oddColor.r(), checkerTexture.oddColor.g(), checkerTexture.oddColor.b(),
            checkerTexture.evenColor.r(), checkerTexture.evenColor.g(), checkerTexture.evenColor.b(),
            checkerTexture.oddTextureName,
            checkerTexture.evenTextureName,
            checkerTexture.scale);

        if (strcmp_device(checkerTexture.name, textureName) == 0)
        {
            return new checker_texture(checkerTexture.scale, checkerTexture.oddColor, checkerTexture.evenColor);
        }
    }

    for (int i = 0; i < sceneCfg->texturesCfg.noiseTextureCount; i++)
    {
        noiseTextureConfig noiseTexture = sceneCfg->texturesCfg.noiseTextures[i];
        printf("[GPU] noiseTexture%d %s %g\n", i,
            noiseTexture.name,
            noiseTexture.scale);

        if (strcmp_device(noiseTexture.name, textureName) == 0)
        {
            return new perlin_noise_texture(noiseTexture.scale);
        }
    }

    for (int i = 0; i < sceneCfg->texturesCfg.bumpTextureCount; i++)
    {
        bumpTextureConfig bumpTexture = sceneCfg->texturesCfg.bumpTextures[i];
        printf("[GPU] bumpTexture%d %s %s %g\n", i,
            bumpTexture.name,
            bumpTexture.filepath,
            bumpTexture.strength);

        if (strcmp_device(bumpTexture.name, textureName) == 0)
        {
            //return new bump_texture(bumpTexture.filepath);
        }
    }

    for (int i = 0; i < sceneCfg->texturesCfg.normalTextureCount; i++)
    {
        normalTextureConfig normalTexture = sceneCfg->texturesCfg.normalTextures[i];
        printf("[GPU] normalTexture%d %s %s %g\n", i,
            normalTexture.name,
            normalTexture.filepath,
            normalTexture.strength);

        if (strcmp_device(normalTexture.name, textureName) == 0)
        {
            //return new normal_texture(bumpTexture.filepath);
        }
    }

    for (int i = 0; i < sceneCfg->texturesCfg.displacementTextureCount; i++)
    {
        displacementTextureConfig displacementTexture = sceneCfg->texturesCfg.displacementTextures[i];
        printf("[GPU] displacementTexture%d %s %s %g\n", i,
            displacementTexture.name,
            displacementTexture.filepath,
            displacementTexture.strength);

        if (strcmp_device(displacementTexture.name, textureName) == 0)
        {
            //return new normal_texture(bumpTexture.filepath);
        }
    }

    for (int i = 0; i < sceneCfg->texturesCfg.alphaTextureCount; i++)
    {
        alphaTextureConfig alphaTexture = sceneCfg->texturesCfg.alphaTextures[i];
        printf("[GPU] alphaTexture%d %s %s %d\n", i,
            alphaTexture.name,
            alphaTexture.filepath,
            alphaTexture.doubleSided);

        if (strcmp_device(alphaTexture.name, textureName) == 0)
        {
            //return new alpha_texture(bumpTexture.filepath);
        }
    }

    for (int i = 0; i < sceneCfg->texturesCfg.emissiveTextureCount; i++)
    {
        emissiveTextureConfig emissiveTexture = sceneCfg->texturesCfg.emissiveTextures[i];
        printf("[GPU] emissiveTexture%d %s %s %g\n", i,
            emissiveTexture.name,
            emissiveTexture.filepath,
            emissiveTexture.strength);

        if (strcmp_device(emissiveTexture.name, textureName) == 0)
        {
            //return new emissive_texture(bumpTexture.filepath);
        }
    }
}


__device__ material* fetchMaterial(sceneConfig* sceneCfg, bitmap_image** images, const char* materialName)
{
    for (int i = 0; i < sceneCfg->materialsCfg.lambertianMaterialCount; i++)
    {
        lambertianMaterialConfig lambertianMaterial = sceneCfg->materialsCfg.lambertianMaterials[i];
        printf("[GPU] lambertianMaterial%d %s %g/%g/%g %s\n", i,
            lambertianMaterial.name,
            lambertianMaterial.rgb.r(), lambertianMaterial.rgb.g(), lambertianMaterial.rgb.b(),
            lambertianMaterial.textureName);

        if (strcmp_device(lambertianMaterial.name, materialName) == 0)
        {
            if (lambertianMaterial.textureName != nullptr && lambertianMaterial.textureName[0] != '\0')
            {
                texture* tex = fetchTexture(sceneCfg, images, lambertianMaterial.textureName);
                if (tex)
                {
                    return new lambertian(tex);
                }
            }
            else
            {
                return new lambertian(lambertianMaterial.rgb);
            }
        }
    }

    for (int i = 0; i < sceneCfg->materialsCfg.metalMaterialCount; i++)
    {
        metalMaterialConfig metalMaterial = sceneCfg->materialsCfg.metalMaterials[i];
        printf("[GPU] metalMaterial%d %s %g/%g/%g %g\n", i,
            metalMaterial.name,
            metalMaterial.rgb.r(), metalMaterial.rgb.g(), metalMaterial.rgb.b(),
            metalMaterial.fuzziness);

        if (strcmp_device(metalMaterial.name, materialName) == 0)
        {
            return new metal(metalMaterial.rgb, metalMaterial.fuzziness);
        }
    }

    for (int i = 0; i < sceneCfg->materialsCfg.dielectricMaterialCount; i++)
    {
        dielectricMaterialConfig glassMaterial = sceneCfg->materialsCfg.dielectricMaterials[i];
        printf("[GPU] glassMaterial%d %s %g\n", i,
            glassMaterial.name,
            glassMaterial.refraction);

        if (strcmp_device(glassMaterial.name, materialName) == 0)
        {
            return new dielectric(glassMaterial.refraction);
        }
    }

    for (int i = 0; i < sceneCfg->materialsCfg.isotropicMaterialCount; i++)
    {
        isotropicMaterialConfig isotropicMaterial = sceneCfg->materialsCfg.isotropicMaterials[i];
        printf("[GPU] isotropicMaterial%d %s %g/%g/%g %s\n", i,
            isotropicMaterial.name,
            isotropicMaterial.rgb.r(), isotropicMaterial.rgb.g(), isotropicMaterial.rgb.b(),
            isotropicMaterial.textureName);

        if (strcmp_device(isotropicMaterial.name, materialName) == 0)
        {
            if (isotropicMaterial.textureName != nullptr && isotropicMaterial.textureName[0] != '\0')
                return new isotropic(isotropicMaterial.rgb);
            else
                return new isotropic(isotropicMaterial.rgb);
        }
    }

    for (int i = 0; i < sceneCfg->materialsCfg.anisotropicMaterialCount; i++)
    {
        anisotropicMaterialConfig anisotropicMaterial = sceneCfg->materialsCfg.anisotropicMaterials[i];
        printf("[GPU] anisotropicMaterial%d %s %g/%g/%g %g %g %s %s %s %g\n", i,
            anisotropicMaterial.name,
            anisotropicMaterial.rgb.r(), anisotropicMaterial.rgb.g(), anisotropicMaterial.rgb.b(),
            anisotropicMaterial.nuf,
            anisotropicMaterial.nvf,
            anisotropicMaterial.diffuseTextureName,
            anisotropicMaterial.specularTextureName,
            anisotropicMaterial.exponentTextureName,
            anisotropicMaterial.roughness);

        if (strcmp_device(anisotropicMaterial.name, materialName) == 0)
        {
            //return new anisotropic(anisotropicMaterial.nuf, anisotropicMaterial.nvf);
        }
    }

    for (int i = 0; i < sceneCfg->materialsCfg.orenNayarMaterialCount; i++)
    {
        orenNayarMaterialConfig orenNayarMaterial = sceneCfg->materialsCfg.orenNayarMaterials[i];
        printf("[GPU] orenNayarMaterial%d %s %g/%g/%g %s %g %g\n", i,
            orenNayarMaterial.name,
            orenNayarMaterial.rgb.r(), orenNayarMaterial.rgb.g(), orenNayarMaterial.rgb.b(),
            orenNayarMaterial.textureName,
            orenNayarMaterial.roughness,
            orenNayarMaterial.albedo_temp);

        if (strcmp_device(orenNayarMaterial.name, materialName) == 0)
        {
            //return oren_nayar();
        }
    }

    for (int i = 0; i < sceneCfg->materialsCfg.phongMaterialCount; i++)
    {
        phongMaterialConfig phongMaterial = sceneCfg->materialsCfg.phongMaterials[i];
        printf("[GPU] phongMaterial%d %s %s %s %s %s %s %s %s %g/%g/%g\n", i,
            phongMaterial.name,
            phongMaterial.diffuseTextureName,
            phongMaterial.specularTextureName,
            phongMaterial.bumpTextureName,
            phongMaterial.normalTextureName,
            phongMaterial.displacementTextureName,
            phongMaterial.alphaTextureName,
            phongMaterial.emissiveTextureName,
            phongMaterial.ambientColor.r(), phongMaterial.ambientColor.g(), phongMaterial.ambientColor.b());

        if (strcmp_device(phongMaterial.name, materialName) == 0)
        {
            //return new phong();
        }
    }
}

__global__ void load_scene(sceneConfig* sceneCfg, hittable_list **elist, hittable_list **elights,  camera **cam, sampler **aa_sampler, int width, int height, float ratio, int spp, int sqrt_spp, bitmap_image** images, int num_textures, int seed)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        // thrust random engine and distribution
        thrust::minstd_rand rng(seed);
        thrust::uniform_real_distribution<float> uniform_dist(0.0f, 1.0f);


        *elights = new hittable_list();

        *elist = new hittable_list();




        // TEXTURES DATA
        // You can access individual textures like this:
        // textures[0] -> first texture
        // textures[1] -> second texture
        // num_textures -> number of textures
        // Example usage:
        //for (int i = 0; i < num_textures; ++i)
        //{
        //    // Process each texture as needed
        //    bitmap_image* current_texture = images[i];
        //    // Use current_texture in the scene (e.g., for material or texture mapping)
        //}



        // LIGHTS
        printf("[GPU] %i omni lights found\n", sceneCfg->lightsCfg.omniLightCount);
        printf("[GPU] %i dir lights found\n", sceneCfg->lightsCfg.dirLightCount);
        printf("[GPU] %i spot lights found\n", sceneCfg->lightsCfg.spotLightCount);

        for (int i = 0; i < sceneCfg->lightsCfg.omniLightCount; i++)
        {
            omniLightConfig omnilight = sceneCfg->lightsCfg.omniLights[i];
            printf("[GPU] omnilight%d %g %s %g/%g/%g %g/%g/%g %g %d\n", i, 
                omnilight.intensity, omnilight.name,
                omnilight.position.x, omnilight.position.y, omnilight.position.z,
                omnilight.rgb.r(), omnilight.rgb.g(), omnilight.rgb.b(),
                omnilight.radius,
                omnilight.invisible);
        }

        for (int i = 0; i < sceneCfg->lightsCfg.dirLightCount; i++)
        {
            directionalLightConfig dirlight = sceneCfg->lightsCfg.dirLights[i];
            printf("[GPU] dirlight%d %g %s %g/%g/%g %g/%g/%g %g/%g/%g %g/%g/%g %d\n", i, 
                dirlight.intensity, dirlight.name,
                dirlight.position.x, dirlight.position.y, dirlight.position.z,
                dirlight.u.x, dirlight.u.y, dirlight.u.z,
                dirlight.v.x, dirlight.v.y, dirlight.v.z,
                dirlight.rgb.r(), dirlight.rgb.g(), dirlight.rgb.b(),
                dirlight.invisible);
        }

        for (int i = 0; i < sceneCfg->lightsCfg.spotLightCount; i++)
        {
            spotLightConfig spotlight = sceneCfg->lightsCfg.spotLights[i];
            printf("[GPU] spotlight%d %g %s %g/%g/%g %g/%g/%g %g %g %g %g/%g/%g %d\n", i, 
                spotlight.intensity, spotlight.name,
                spotlight.position.x, spotlight.position.y, spotlight.position.z,
                spotlight.direction.x, spotlight.direction.y, spotlight.direction.z,
                spotlight.cutoff,
                spotlight.falloff,
                spotlight.radius,
                spotlight.rgb.r(), spotlight.rgb.g(), spotlight.rgb.b(),
                spotlight.invisible);
        }


        // PRIMITIVES
        for (int i = 0; i < sceneCfg->primitivesCfg.spherePrimitiveCount; i++)
        {
            spherePrimitiveConfig spherePrimitive = sceneCfg->primitivesCfg.spherePrimitives[i];

            material* mat = fetchMaterial(sceneCfg, images, spherePrimitive.materialName);

            if (mat)
                (*elist)->add(scene_factory::createSphere(spherePrimitive.name, spherePrimitive.position, spherePrimitive.radius, mat, spherePrimitive.mapping));

            printf("[GPU] spherePrimitive%d %s %g/%g/%g %g/%g %g/%g %g/%g %g %s %s\n", i,
                spherePrimitive.name,
                spherePrimitive.position.x, spherePrimitive.position.y, spherePrimitive.position.z,
                spherePrimitive.mapping.offset_u(), spherePrimitive.mapping.offset_v(),
                spherePrimitive.mapping.repeat_u(), spherePrimitive.mapping.repeat_v(),
                spherePrimitive.mapping.scale_u(), spherePrimitive.mapping.scale_v(),
                spherePrimitive.radius,
                spherePrimitive.materialName,
                spherePrimitive.groupName);
        }

        for (int i = 0; i < sceneCfg->primitivesCfg.planePrimitiveCount; i++)
        {
            planePrimitiveConfig planePrimitive = sceneCfg->primitivesCfg.planePrimitives[i];

            material* mat = fetchMaterial(sceneCfg, images, planePrimitive.materialName);

            if (mat)
                (*elist)->add(scene_factory::createPlane(planePrimitive.name, planePrimitive.point1, planePrimitive.point2, mat, planePrimitive.mapping));

            printf("[GPU] planePrimitive%d %s %g/%g/%g %g/%g/%g %g/%g %g/%g %g/%g %s %s\n", i,
                planePrimitive.name,
                planePrimitive.point1.x, planePrimitive.point1.y, planePrimitive.point1.z,
                planePrimitive.point2.x, planePrimitive.point2.y, planePrimitive.point2.z,
                planePrimitive.mapping.offset_u(), planePrimitive.mapping.offset_v(),
                planePrimitive.mapping.repeat_u(), planePrimitive.mapping.repeat_v(),
                planePrimitive.mapping.scale_u(), planePrimitive.mapping.scale_v(),
                planePrimitive.materialName,
                planePrimitive.groupName);
        }

        for (int i = 0; i < sceneCfg->primitivesCfg.quadPrimitiveCount; i++)
        {
            quadPrimitiveConfig quadPrimitive = sceneCfg->primitivesCfg.quadPrimitives[i];
            
            material* mat = fetchMaterial(sceneCfg, images, quadPrimitive.materialName);
            
            if (mat)
                (*elist)->add(scene_factory::createQuad(quadPrimitive.name, quadPrimitive.position, quadPrimitive.u, quadPrimitive.v, mat, quadPrimitive.mapping));

            printf("[GPU] quadPrimitive%d %s %g/%g/%g %g/%g/%g %g/%g/%g %g/%g %g/%g %g/%g %s %s\n", i,
                quadPrimitive.name,
                quadPrimitive.position.x, quadPrimitive.position.y, quadPrimitive.position.z,
                quadPrimitive.u.x, quadPrimitive.u.y, quadPrimitive.u.z,
                quadPrimitive.v.x, quadPrimitive.v.y, quadPrimitive.v.z,
                quadPrimitive.mapping.offset_u(), quadPrimitive.mapping.offset_v(),
                quadPrimitive.mapping.repeat_u(), quadPrimitive.mapping.repeat_v(),
                quadPrimitive.mapping.scale_u(), quadPrimitive.mapping.scale_v(),
                quadPrimitive.materialName,
                quadPrimitive.groupName);
        }

        for (int i = 0; i < sceneCfg->primitivesCfg.boxPrimitiveCount; i++)
        {
            boxPrimitiveConfig boxPrimitive = sceneCfg->primitivesCfg.boxPrimitives[i];

            material* mat = fetchMaterial(sceneCfg, images, boxPrimitive.materialName);

            if (mat)
                (*elist)->add(scene_factory::createBox(boxPrimitive.name, boxPrimitive.position, boxPrimitive.size, mat, boxPrimitive.mapping));


            printf("[GPU] boxPrimitive%d %s %g/%g/%g %g/%g/%g %g/%g %g/%g %g/%g %s %s\n", i,
                boxPrimitive.name,
                boxPrimitive.position.x, boxPrimitive.position.y, boxPrimitive.position.z,
                boxPrimitive.size.x, boxPrimitive.size.y, boxPrimitive.size.z,
                boxPrimitive.mapping.offset_u(), boxPrimitive.mapping.offset_v(),
                boxPrimitive.mapping.repeat_u(), boxPrimitive.mapping.repeat_v(),
                boxPrimitive.mapping.scale_u(), boxPrimitive.mapping.scale_v(),
                boxPrimitive.materialName,
                boxPrimitive.groupName);
        }

        for (int i = 0; i < sceneCfg->primitivesCfg.conePrimitiveCount; i++)
        {
            conePrimitiveConfig conePrimitive = sceneCfg->primitivesCfg.conePrimitives[i];
            printf("[GPU] conePrimitive%d %s %g/%g/%g %g %g %g/%g %g/%g %g/%g %s %s\n", i,
                conePrimitive.name,
                conePrimitive.position.x, conePrimitive.position.y, conePrimitive.position.z,
                conePrimitive.radius,
                conePrimitive.height,
                conePrimitive.mapping.offset_u(), conePrimitive.mapping.offset_v(),
                conePrimitive.mapping.repeat_u(), conePrimitive.mapping.repeat_v(),
                conePrimitive.mapping.scale_u(), conePrimitive.mapping.scale_v(),
                conePrimitive.materialName,
                conePrimitive.groupName);
        }

        for (int i = 0; i < sceneCfg->primitivesCfg.cylinderPrimitiveCount; i++)
        {
            cylinderPrimitiveConfig cylinderPrimitive = sceneCfg->primitivesCfg.cylinderPrimitives[i];
            printf("[GPU] cylinderPrimitive%d %s %g/%g/%g %g %g %g/%g %g/%g %g/%g %s %s\n", i,
                cylinderPrimitive.name,
                cylinderPrimitive.position.x, cylinderPrimitive.position.y, cylinderPrimitive.position.z,
                cylinderPrimitive.radius,
                cylinderPrimitive.height,
                cylinderPrimitive.mapping.offset_u(), cylinderPrimitive.mapping.offset_v(),
                cylinderPrimitive.mapping.repeat_u(), cylinderPrimitive.mapping.repeat_v(),
                cylinderPrimitive.mapping.scale_u(), cylinderPrimitive.mapping.scale_v(),
                cylinderPrimitive.materialName,
                cylinderPrimitive.groupName);
        }

        for (int i = 0; i < sceneCfg->primitivesCfg.diskPrimitiveCount; i++)
        {
            diskPrimitiveConfig diskPrimitive = sceneCfg->primitivesCfg.diskPrimitives[i];
            printf("[GPU] diskPrimitive%d %s %g/%g/%g %g %g %g/%g %g/%g %g/%g %s %s\n", i,
                diskPrimitive.name,
                diskPrimitive.position.x, diskPrimitive.position.y, diskPrimitive.position.z,
                diskPrimitive.radius,
                diskPrimitive.height,
                diskPrimitive.mapping.offset_u(), diskPrimitive.mapping.offset_v(),
                diskPrimitive.mapping.repeat_u(), diskPrimitive.mapping.repeat_v(),
                diskPrimitive.mapping.scale_u(), diskPrimitive.mapping.scale_v(),
                diskPrimitive.materialName,
                diskPrimitive.groupName);
        }

        for (int i = 0; i < sceneCfg->primitivesCfg.torusPrimitiveCount; i++)
        {
            torusPrimitiveConfig torusPrimitive = sceneCfg->primitivesCfg.torusPrimitives[i];
            printf("[GPU] diskPrimitive%d %s %g/%g/%g %g %g %g/%g %g/%g %g/%g %s %s\n", i,
                torusPrimitive.name,
                torusPrimitive.position.x, torusPrimitive.position.y, torusPrimitive.position.z,
                torusPrimitive.minor_radius,
                torusPrimitive.major_radius,
                torusPrimitive.mapping.offset_u(), torusPrimitive.mapping.offset_v(),
                torusPrimitive.mapping.repeat_u(), torusPrimitive.mapping.repeat_v(),
                torusPrimitive.mapping.scale_u(), torusPrimitive.mapping.scale_v(),
                torusPrimitive.materialName,
                torusPrimitive.groupName);
        }

        for (int i = 0; i < sceneCfg->primitivesCfg.volumePrimitiveCount; i++)
        {
            volumePrimitiveConfig volumePrimitive = sceneCfg->primitivesCfg.volumePrimitives[i];
            printf("[GPU] volumePrimitive%d %s %s %g %g/%g/%g %s %s\n", i,
                volumePrimitive.name,
                volumePrimitive.boundaryName,
                volumePrimitive.density,
                volumePrimitive.rgb.r(), volumePrimitive.rgb.g(), volumePrimitive.rgb.b(),
                volumePrimitive.textureName,
                volumePrimitive.groupName);
        }



        //hittable_list obj = composer->getObjects();
        //for (int w = 0; w < obj.object_count; w++)
        //{
        //    auto sss = obj.objects[w];
        //    (*elist)->add(sss);
        //}


        




        //(*elist)->add(new rt::flip_normals(new yz_rect(0, 555, 0, 555, 555, new lambertian(new solid_color_texture(color(0.12, 0.45, 0.15))), "MyLeft")));
        //(*elist)->add(new yz_rect(0, 555, 0, 555, 0, new lambertian(new solid_color_texture(color(0.65, 0.05, 0.05))), "MyRight"));
        //(*elist)->add(new xz_rect(0, 555, 0, 555, 0, new lambertian(new solid_color_texture(color(0.73, 0.73, 0.73))), "MyGround"));
        //(*elist)->add(new rt::flip_normals(new xz_rect(0, 555, 0, 555, 555, new lambertian(new solid_color_texture(color(0.73, 0.73, 0.73))), "MyTop")));
        //(*elist)->add(new rt::flip_normals(new xz_rect(0, 555, 0, 555, 555, new lambertian(new solid_color_texture(color(0.73, 0.73, 0.73))), "MyBottom")));
        
        // back
        //(*elist)->add(new quad(point3(0,0,555), vector3(555,0,0), vector3(0,555,0), new lambertian(new solid_color_texture(color(0.73, 0.73, 0.73))), "MyBack"));


        // box
        (*elist)->add(new rt::translate(new box(point3(0.0f, 0.0f, 200.0f), vector3(165, 330, 165), new lambertian(new image_texture(*(images[0]))), "MyBox"), vector3(120,0,320)));
        
        // sphere
        //(*elist)->add(new sphere(point3(350.0f, 50.0f, 295.0f), 100.0f, new lambertian(*texture), "MySphere"));


        // light
        (*elist)->add(new directional_light(point3(278, 554, 332), vector3(-305, 0, 0), vector3(0, 0, -305), 1.0f, color(10.0, 10.0, 10.0), "MyLight", true));




        // temp extract_emissive_objects
        for (int i = 0; i < (*elist)->object_count; i++)
        {
            if ((*elist)->objects[i]->getTypeID() == HittableTypeID::lightDirectionalType)
            {
                light* derived = static_cast<light*>((*elist)->objects[i]);
                if (derived)
                {
                    (*elights)->add((*elist)->objects[i]);
                }
            }
        }

        *cam = new perspective_camera();
        (*cam)->initialize(
            vector3(278, 278, -800),
            vector3(278, 278, 0),
            vector3(0, 1, 0),
            width,
            ratio,
            40.0f,
            0.0f,
            10.0f,
            0.0f,
            0.0f,
            1.0f,
            sqrt_spp);


        // calculate bounding boxes to speed up ray computing
        *elist = new hittable_list(new bvh_node((*elist)->objects, 0, (*elist)->object_count, rng));
    }
}

__global__ void texture_init(unsigned char* tex_data, int width, int height, int channels, bitmap_image** tex)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        *tex = new bitmap_image(tex_data, width, height, channels);
    }
}

__global__ void render(scene* world_scene, color* fb, int width, int height, int spp, int sqrt_spp, int max_depth, hittable_list** world, hittable_list** lights, camera** cam, sampler** aa_sampler, int seed)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= width) || (j >= height)) return;

    int pixel_index = j * width + i;

    // Initialize the random engine and distribution
    thrust::minstd_rand rng(seed + pixel_index);
    thrust::uniform_real_distribution<float> uniform_dist(0.0f, 1.0f);


    color pixel_color(0, 0, 0);

    for (int s_j = 0; s_j < sqrt_spp; ++s_j)
    {
        for (int s_i = 0; s_i < sqrt_spp; ++s_i)
        {
            // Generate a random value between 0 and 1
            float uniform_random = uniform_dist(rng);

            // Stratified sampling within the pixel, with Sobol randomness
            float u = (i + (s_i + uniform_random) / sqrt_spp) / float(width);
            float v = (j + (s_j + uniform_random) / sqrt_spp) / float(height);

            ray r = (*cam)->get_ray(u, v, s_i, s_j, nullptr, rng);
            pixel_color += (*cam)->ray_color(r, i, j, max_depth, max_depth, **world, **lights, rng);
        }
    }

    const color& fix = prepare_pixel_color(i, j, pixel_color, spp, true);
    const interval intensity(0.000f, 0.999f);

    int color_r = static_cast<int>(255.99f * intensity.clamp(fix.r()));
    int color_g = static_cast<int>(255.99f * intensity.clamp(fix.g()));
    int color_b = static_cast<int>(255.99f * intensity.clamp(fix.b()));

    fb[pixel_index] = color(color_r, color_g, color_b);

    printf("p %u %u %u %u %u\n", i, height - j - 1, color_r, color_g, color_b);
}


void setupCuda(const hipDeviceProp_t& prop)
{
    // If you get a null pointer (either from device malloc or device new) you have run out of heap space.
    // https://forums.developer.nvidia.com/t/allocating-memory-from-device-and-cudalimitmallocheapsize/70441
    
    size_t stackSize;

    // Get the current stack size limit
    hipError_t result1 = hipDeviceGetLimit(&stackSize, hipLimitStackSize);
    if (result1 != hipSuccess) {
        std::cerr << "[WARNING] Failed to get stack size: " << hipGetErrorString(result1) << std::endl;
        return;
    }

    std::cout << "[INFO] Current stack size limit: " << stackSize << " bytes" << std::endl;


    const size_t newStackSize = 4096; // Set the stack size to 1MB per thread

    hipError_t result2 = hipDeviceSetLimit(hipLimitStackSize, newStackSize);
    if (result2 != hipSuccess) {
        std::cerr << "[WARNING] Failed to set stack size: " << hipGetErrorString(result2) << std::endl;
        return;
    }

    std::cout << "[INFO] New stack size limit: " << newStackSize << " bytes" << std::endl;



    const size_t newMallocHeapSize = size_t(1024) * size_t(1024) * size_t(1024);

    hipError_t result3 = hipDeviceSetLimit(hipLimitMallocHeapSize, newMallocHeapSize);
    if (result3 != hipSuccess) {
        std::cerr << "[WARNING] Failed to set malloc heap size: " << hipGetErrorString(result3) << std::endl;
        return;
    }

    std::cout << "[INFO] New malloc heap limit: " << newMallocHeapSize << " bytes" << std::endl;



    const size_t newPrintfFifoSize = 10000000;

    hipError_t result4 = hipDeviceSetLimit(hipLimitPrintfFifoSize, newPrintfFifoSize);
    if (result4 != hipSuccess) {
        std::cerr << "[WARNING] Failed to set printf fifo size: " << hipGetErrorString(result4) << std::endl;
        return;
    }

    std::cout << "[INFO] New printf fifo size: " << newPrintfFifoSize << " bytes" << std::endl;
}

/// <summary>
/// Helper function to copy a string from the host to the device
/// </summary>
/// <param name="hostString">Pointer to the host string</param>
/// <param name="deviceString">Pointer to the device string (output)</param>
void copyStringToDevice(const char* hostString, char** deviceString)
{
    // Allocate memory on the device for the string (with null terminator)
    size_t stringLen = strlen(hostString) + 1;  // +1 for null terminator
    hipMalloc((void**)deviceString, stringLen);

    // Copy the string from host to device
    hipMemcpy(*deviceString, hostString, stringLen, hipMemcpyHostToDevice);
}


/// <summary>
/// // Helper function to copy texture configuration
/// </summary>
template<typename TextureConfig>
void copyCommonTextureConfig(const TextureConfig* h_textures, int count, TextureConfig** d_textures, texturesConfig* d_texturesCfg, TextureConfig** d_texturesPtrOnDevice)
{
    // 1. Allocate memory for the array on the device
    hipMalloc((void**)d_textures, count * sizeof(TextureConfig));

    // 2. Copy the array contents from host to device
    hipMemcpy(*d_textures, h_textures, count * sizeof(TextureConfig), hipMemcpyHostToDevice);

    // 3. Allocate memory and copy the names for each texture
    for (int i = 0; i < count; i++)
    {
        // Copy name
        const char* hostName = h_textures[i].name;  // Get the string from the host
        char* d_name;
        copyStringToDevice(hostName, &d_name);  // Use reusable function for name
        hipMemcpy(&((*d_textures)[i].name), &d_name, sizeof(char*), hipMemcpyHostToDevice);
    }

    // 4. Update the device-side config to point to the array on the device
    hipMemcpy(d_texturesPtrOnDevice, d_textures, sizeof(TextureConfig*), hipMemcpyHostToDevice);
}




/// <summary>
/// // Helper function to copy common material configuration
/// </summary>
template<typename MaterialConfig>
void copyCommonMaterialConfig(const MaterialConfig* h_materials, int count, MaterialConfig** d_materials, materialsConfig* d_materialsCfg, MaterialConfig** d_materialsPtrOnDevice)
{
    // 1. Allocate memory for the array on the device
    hipMalloc((void**)d_materials, count * sizeof(MaterialConfig));

    // 2. Copy the array contents from host to device
    hipMemcpy(*d_materials, h_materials, count * sizeof(MaterialConfig), hipMemcpyHostToDevice);

    // 3. Allocate memory and copy the names for each texture
    for (int i = 0; i < count; i++)
    {
        // Copy name
        const char* hostName = h_materials[i].name;  // Get the string from the host
        char* d_name;
        copyStringToDevice(hostName, &d_name);  // Use reusable function for name
        hipMemcpy(&((*d_materials)[i].name), &d_name, sizeof(char*), hipMemcpyHostToDevice);
    }

    // 4. Update the device-side config to point to the array on the device
    hipMemcpy(d_materialsPtrOnDevice, d_materials, sizeof(MaterialConfig*), hipMemcpyHostToDevice);
}

/// <summary>
/// // Helper function to copy common material configuration
/// </summary>
template<typename MaterialConfig>
void copyTextureMaterialConfig(const MaterialConfig* h_materials, int count, MaterialConfig** d_materials, materialsConfig* d_materialsCfg, MaterialConfig** d_materialsPtrOnDevice)
{
    // 1. Allocate memory for the array on the device
    hipMalloc((void**)d_materials, count * sizeof(MaterialConfig));

    // 2. Copy the array contents from host to device
    hipMemcpy(*d_materials, h_materials, count * sizeof(MaterialConfig), hipMemcpyHostToDevice);

    // 3. Allocate memory and copy the names for each texture
    for (int i = 0; i < count; i++)
    {
        // Copy name
        const char* hostName = h_materials[i].name;  // Get the string from the host
        char* d_name;
        copyStringToDevice(hostName, &d_name);  // Use reusable function for name
        hipMemcpy(&((*d_materials)[i].name), &d_name, sizeof(char*), hipMemcpyHostToDevice);

        // Copy textureName
        const char* hostTextureName = h_materials[i].textureName;  // Get the string from the host
        char* d_textureName;
        copyStringToDevice(hostTextureName, &d_textureName);  // Use reusable function for name
        hipMemcpy(&((*d_materials)[i].textureName), &d_textureName, sizeof(char*), hipMemcpyHostToDevice);
    }

    // 4. Update the device-side config to point to the array on the device
    hipMemcpy(d_materialsPtrOnDevice, d_materials, sizeof(MaterialConfig*), hipMemcpyHostToDevice);
}


/// <summary>
/// // Helper function to copy anisotropic material configuration
/// </summary>
template<typename MaterialConfig>
void copyAnisotropicMaterialConfig(const MaterialConfig* h_materials, int count, MaterialConfig** d_materials, materialsConfig* d_materialsCfg, MaterialConfig** d_materialsPtrOnDevice)
{
    // 1. Allocate memory for the array on the device
    hipMalloc((void**)d_materials, count * sizeof(MaterialConfig));

    // 2. Copy the array contents from host to device
    hipMemcpy(*d_materials, h_materials, count * sizeof(MaterialConfig), hipMemcpyHostToDevice);

    // 3. Allocate memory and copy the names for each texture
    for (int i = 0; i < count; i++)
    {
        // Copy name
        const char* hostName = h_materials[i].name;  // Get the string from the host
        char* d_name;
        copyStringToDevice(hostName, &d_name);  // Use reusable function for name
        hipMemcpy(&((*d_materials)[i].name), &d_name, sizeof(char*), hipMemcpyHostToDevice);

        // Copy diffuseTextureName
        const char* hostDiffuseTextureName = h_materials[i].diffuseTextureName;  // Get the string from the host
        char* d_diffuseTextureName;
        copyStringToDevice(hostDiffuseTextureName, &d_diffuseTextureName);  // Use reusable function for name
        hipMemcpy(&((*d_materials)[i].diffuseTextureName), &d_diffuseTextureName, sizeof(char*), hipMemcpyHostToDevice);

        // Copy diffuseTextureName
        const char* hostSpecularTextureName = h_materials[i].specularTextureName;  // Get the string from the host
        char* d_specularTextureName;
        copyStringToDevice(hostSpecularTextureName, &d_specularTextureName);  // Use reusable function for name
        hipMemcpy(&((*d_materials)[i].specularTextureName), &d_specularTextureName, sizeof(char*), hipMemcpyHostToDevice);

        // Copy exponentTextureName
        const char* hostExponentTextureName = h_materials[i].exponentTextureName;  // Get the string from the host
        char* d_exponentTextureName;
        copyStringToDevice(hostExponentTextureName, &d_exponentTextureName);  // Use reusable function for name
        hipMemcpy(&((*d_materials)[i].exponentTextureName), &d_exponentTextureName, sizeof(char*), hipMemcpyHostToDevice);
    }

    // 4. Update the device-side config to point to the array on the device
    hipMemcpy(d_materialsPtrOnDevice, d_materials, sizeof(MaterialConfig*), hipMemcpyHostToDevice);
}

/// <summary>
/// // Helper function to copy phong material configuration
/// </summary>
template<typename MaterialConfig>
void copyPhongMaterialConfig(const MaterialConfig* h_materials, int count, MaterialConfig** d_materials, materialsConfig* d_materialsCfg, MaterialConfig** d_materialsPtrOnDevice)
{
    // 1. Allocate memory for the array on the device
    hipMalloc((void**)d_materials, count * sizeof(MaterialConfig));

    // 2. Copy the array contents from host to device
    hipMemcpy(*d_materials, h_materials, count * sizeof(MaterialConfig), hipMemcpyHostToDevice);

    // 3. Allocate memory and copy the names for each texture
    for (int i = 0; i < count; i++)
    {
        // Copy name
        const char* hostName = h_materials[i].name;  // Get the string from the host
        char* d_name;
        copyStringToDevice(hostName, &d_name);  // Use reusable function for name
        hipMemcpy(&((*d_materials)[i].name), &d_name, sizeof(char*), hipMemcpyHostToDevice);

        // Copy diffuseTextureName
        const char* hostDiffuseTextureName = h_materials[i].diffuseTextureName;  // Get the string from the host
        char* d_diffuseTextureName;
        copyStringToDevice(hostDiffuseTextureName, &d_diffuseTextureName);  // Use reusable function
        hipMemcpy(&((*d_materials)[i].diffuseTextureName), &d_diffuseTextureName, sizeof(char*), hipMemcpyHostToDevice);

        // Copy specularTextureName
        const char* hostSpecularTextureName = h_materials[i].specularTextureName;  // Get the string from the host
        char* d_specularTextureName;
        copyStringToDevice(hostSpecularTextureName, &d_specularTextureName);  // Use reusable function
        hipMemcpy(&((*d_materials)[i].specularTextureName), &d_specularTextureName, sizeof(char*), hipMemcpyHostToDevice);

        // Copy bumpTextureName
        const char* hostBumpTextureName = h_materials[i].bumpTextureName;  // Get the string from the host
        char* d_bumpTextureName;
        copyStringToDevice(hostBumpTextureName, &d_bumpTextureName);  // Use reusable function
        hipMemcpy(&((*d_materials)[i].bumpTextureName), &d_bumpTextureName, sizeof(char*), hipMemcpyHostToDevice);

        // Copy normalTextureName
        const char* hostNormalTextureName = h_materials[i].normalTextureName;  // Get the string from the host
        char* d_normalTextureName;
        copyStringToDevice(hostNormalTextureName, &d_normalTextureName);  // Use reusable function
        hipMemcpy(&((*d_materials)[i].normalTextureName), &d_normalTextureName, sizeof(char*), hipMemcpyHostToDevice);

        // Copy displacementTextureName
        const char* hostDisplacementTextureName = h_materials[i].displacementTextureName;  // Get the string from the host
        char* d_displacementTextureName;
        copyStringToDevice(hostDisplacementTextureName, &d_displacementTextureName);  // Use reusable function
        hipMemcpy(&((*d_materials)[i].displacementTextureName), &d_displacementTextureName, sizeof(char*), hipMemcpyHostToDevice);

        // Copy alphaTextureName
        const char* hostAlphaTextureName = h_materials[i].alphaTextureName;  // Get the string from the host
        char* d_alphaTextureName;
        copyStringToDevice(hostAlphaTextureName, &d_alphaTextureName);  // Use reusable function
        hipMemcpy(&((*d_materials)[i].alphaTextureName), &d_alphaTextureName, sizeof(char*), hipMemcpyHostToDevice);


        // Copy emissiveTextureName
        const char* hostEmissiveTextureName = h_materials[i].emissiveTextureName;  // Get the string from the host
        char* d_emissiveTextureName;
        copyStringToDevice(hostEmissiveTextureName, &d_emissiveTextureName);  // Use reusable function
        hipMemcpy(&((*d_materials)[i].emissiveTextureName), &d_emissiveTextureName, sizeof(char*), hipMemcpyHostToDevice);
    }

    // 4. Update the device-side config to point to the array on the device
    hipMemcpy(d_materialsPtrOnDevice, d_materials, sizeof(MaterialConfig*), hipMemcpyHostToDevice);
}




/// <summary>
/// // Helper function to copy common texture configuration
/// </summary>
template<typename TextureConfig>
void copyImageTextureConfig(const TextureConfig* h_textures, int count, TextureConfig** d_textures, texturesConfig* d_texturesCfg, TextureConfig** d_texturesPtrOnDevice)
{
    // 1. Allocate memory for the array on the device
    hipMalloc((void**)d_textures, count * sizeof(TextureConfig));

    // 2. Copy the array contents from host to device
    hipMemcpy(*d_textures, h_textures, count * sizeof(TextureConfig), hipMemcpyHostToDevice);

    // 3. Allocate memory and copy the names for each texture
    for (int i = 0; i < count; i++)
    {
        // Copy name
        const char* hostName = h_textures[i].name;  // Get the string from the host
        char* d_name;
        copyStringToDevice(hostName, &d_name);  // Use reusable function for name
        hipMemcpy(&((*d_textures)[i].name), &d_name, sizeof(char*), hipMemcpyHostToDevice);

        // Copy filepath
        const char* hostFilepath = h_textures[i].filepath;  // Get the string from the host
        char* d_filepath;
        copyStringToDevice(hostFilepath, &d_filepath);  // Use reusable function for filepath
        hipMemcpy(&((*d_textures)[i].filepath), &d_filepath, sizeof(char*), hipMemcpyHostToDevice);
    }

    // 4. Update the device-side config to point to the array on the device
    hipMemcpy(d_texturesPtrOnDevice, d_textures, sizeof(TextureConfig*), hipMemcpyHostToDevice);
}

/// <summary>
/// // Helper function to copy common primitive configuration
/// </summary>
template<typename PrimitiveConfig>
void copyCommonPrimitiveConfig(const PrimitiveConfig* h_primitives, int count, PrimitiveConfig** d_primitives, primitivesConfig* d_primitivesCfg, PrimitiveConfig** d_primitivesPtrOnDevice)
{
    // 1. Allocate memory for the array on the device
    hipMalloc((void**)d_primitives, count * sizeof(PrimitiveConfig));

    // 2. Copy the array contents from host to device
    hipMemcpy(*d_primitives, h_primitives, count * sizeof(PrimitiveConfig), hipMemcpyHostToDevice);

    // 3. Allocate memory and copy the names for each primitive
    for (int i = 0; i < count; i++)
    {
        // Copy name
        const char* hostName = h_primitives[i].name;  // Get the string from the host
        char* d_name;
        copyStringToDevice(hostName, &d_name);  // Use reusable function for name
        hipMemcpy(&((*d_primitives)[i].name), &d_name, sizeof(char*), hipMemcpyHostToDevice);

        // Copy materialName
        const char* hostMaterialName = h_primitives[i].materialName;  // Get the string from the host
        char* d_materialName;
        copyStringToDevice(hostMaterialName, &d_materialName);  // Use reusable function for material name
        hipMemcpy(&((*d_primitives)[i].materialName), &d_materialName, sizeof(char*), hipMemcpyHostToDevice);

        // Copy groupName
        const char* hostGroupName = h_primitives[i].groupName;  // Get the string from the host
        char* d_groupName;
        copyStringToDevice(hostGroupName, &d_groupName);  // Use reusable function for group name
        hipMemcpy(&((*d_primitives)[i].groupName), &d_groupName, sizeof(char*), hipMemcpyHostToDevice);
    }

    // 4. Update the device-side config to point to the array on the device
    hipMemcpy(d_primitivesPtrOnDevice, d_primitives, sizeof(PrimitiveConfig*), hipMemcpyHostToDevice);
}



/// <summary>
/// // Helper function to copy volume primitive configuration
/// </summary>
template<typename PrimitiveConfig>
void copyVolumePrimitiveConfig(const PrimitiveConfig* h_primitives, int count, PrimitiveConfig** d_primitives, primitivesConfig* d_primitivesCfg, PrimitiveConfig** d_primitivesPtrOnDevice)
{
    // 1. Allocate memory for the array on the device
    hipMalloc((void**)d_primitives, count * sizeof(PrimitiveConfig));

    // 2. Copy the array contents from host to device
    hipMemcpy(*d_primitives, h_primitives, count * sizeof(PrimitiveConfig), hipMemcpyHostToDevice);

    // 3. Allocate memory and copy the names for each primitive
    for (int i = 0; i < count; i++)
    {
        // Copy name
        const char* hostName = h_primitives[i].name;  // Get the string from the host
        char* d_name;
        copyStringToDevice(hostName, &d_name);  // Use reusable function for name
        hipMemcpy(&((*d_primitives)[i].name), &d_name, sizeof(char*), hipMemcpyHostToDevice);

        // Copy boundaryObjectName
        const char* hostBoundaryName = h_primitives[i].boundaryName;  // Get the string from the host
        char* d_boundaryName;
        copyStringToDevice(hostBoundaryName, &d_boundaryName);  // Use reusable function for boundary name
        hipMemcpy(&((*d_primitives)[i].boundaryName), &d_boundaryName, sizeof(char*), hipMemcpyHostToDevice);

        // Copy textureName
        const char* hostTextureName = h_primitives[i].textureName;  // Get the string from the host
        char* d_textureName;
        copyStringToDevice(hostTextureName, &d_textureName);  // Use reusable function for texture name
        hipMemcpy(&((*d_primitives)[i].textureName), &d_textureName, sizeof(char*), hipMemcpyHostToDevice);

        // Copy groupName
        const char* hostGroupName = h_primitives[i].groupName;  // Get the string from the host
        char* d_groupName;
        copyStringToDevice(hostGroupName, &d_groupName);  // Use reusable function for group name
        hipMemcpy(&((*d_primitives)[i].groupName), &d_groupName, sizeof(char*), hipMemcpyHostToDevice);
    }

    // 4. Update the device-side config to point to the array on the device
    hipMemcpy(d_primitivesPtrOnDevice, d_primitives, sizeof(PrimitiveConfig*), hipMemcpyHostToDevice);
}


/// <summary>
/// // Helper function to copy texture configuration
/// </summary>
template<typename TextureConfig>
void copyCheckerTextureConfig(const TextureConfig* h_textures, int count, TextureConfig** d_textures, texturesConfig* d_texturesCfg, TextureConfig** d_texturesPtrOnDevice)
{
    // 1. Allocate memory for the array on the device
    hipMalloc((void**)d_textures, count * sizeof(TextureConfig));

    // 2. Copy the array contents from host to device
    hipMemcpy(*d_textures, h_textures, count * sizeof(TextureConfig), hipMemcpyHostToDevice);

    // 3. Allocate memory and copy the names for each texture
    for (int i = 0; i < count; i++)
    {
        // Copy name
        const char* hostName = h_textures[i].name;  // Get the string from the host
        char* d_name;
        copyStringToDevice(hostName, &d_name);  // Use reusable function for name
        hipMemcpy(&((*d_textures)[i].name), &d_name, sizeof(char*), hipMemcpyHostToDevice);

        // Copy oddTextureName
        const char* hostOddTextureName = h_textures[i].oddTextureName;  // Get the string from the host
        char* d_oddTextureName;
        copyStringToDevice(hostOddTextureName, &d_oddTextureName);  // Use reusable function for filepath
        hipMemcpy(&((*d_textures)[i].oddTextureName), &d_oddTextureName, sizeof(char*), hipMemcpyHostToDevice);

        // Copy evenTextureName
        const char* hostEvenTextureName = h_textures[i].evenTextureName;  // Get the string from the host
        char* d_evenTextureName;
        copyStringToDevice(hostEvenTextureName, &d_evenTextureName);  // Use reusable function for filepath
        hipMemcpy(&((*d_textures)[i].evenTextureName), &d_evenTextureName, sizeof(char*), hipMemcpyHostToDevice);
    }

    // 4. Update the device-side config to point to the array on the device
    hipMemcpy(d_texturesPtrOnDevice, d_textures, sizeof(TextureConfig*), hipMemcpyHostToDevice);
}


/// <summary>
/// Helper function to copy light configuration
/// </summary>
template<typename LightConfig>
void copyLightConfig(const LightConfig* h_lights, int count, LightConfig** d_lights, lightsConfig* d_lightsCfg, LightConfig** d_lightsPtrOnDevice)
{
    // 1. Allocate memory for the lights array on the device
    hipMalloc((void**)d_lights, count * sizeof(LightConfig));

    // 2. Copy the lights array contents from host to device
    hipMemcpy(*d_lights, h_lights, count * sizeof(LightConfig), hipMemcpyHostToDevice);

    // 3. Allocate memory and copy the names for each light
    for (int i = 0; i < count; i++)
    {
        // Copy name
        const char* hostName = h_lights[i].name;  // Get the string from the host
        char* d_name;
        copyStringToDevice(hostName, &d_name);  // Use reusable function for name
        hipMemcpy(&((*d_lights)[i].name), &d_name, sizeof(char*), hipMemcpyHostToDevice);
    }

    // 4. Update the device-side lightsConfig to point to the lights array on the device
    hipMemcpy(d_lightsPtrOnDevice, d_lights, sizeof(LightConfig*), hipMemcpyHostToDevice);
}



texturesConfig* prepareTextures(const texturesConfig& h_texturesCfg)
{
    // Allocate and copy the textures data (for solid color, gradient color, image...)
    texturesConfig* d_texturesCfg;
    hipMalloc((void**)&d_texturesCfg, sizeof(texturesConfig));

    // Solid color textures
    if (h_texturesCfg.solidColorTextureCount > 0)
    {
        solidColorTextureConfig* d_solidColorTextures;
        copyCommonTextureConfig(h_texturesCfg.solidColorTextures, h_texturesCfg.solidColorTextureCount, &d_solidColorTextures, d_texturesCfg, &(d_texturesCfg->solidColorTextures));
    }

    // Copy the scalar values from host to device for solid color texture count
    hipMemcpy(&(d_texturesCfg->solidColorTextureCount), &(h_texturesCfg.solidColorTextureCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Gradient color textures
    if (h_texturesCfg.gradientColorTextureCount > 0)
    {
        gradientColorTextureConfig* d_gradientColorTextures;
        copyCommonTextureConfig(h_texturesCfg.gradientColorTextures, h_texturesCfg.gradientColorTextureCount, &d_gradientColorTextures, d_texturesCfg, &(d_texturesCfg->gradientColorTextures));
    }

    // Copy the scalar values from host to device for gradient color texture count
    hipMemcpy(&(d_texturesCfg->gradientColorTextureCount), &(h_texturesCfg.gradientColorTextureCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Image textures
    if (h_texturesCfg.imageTextureCount > 0)
    {
        imageTextureConfig* d_imageTextures;
        copyImageTextureConfig(h_texturesCfg.imageTextures, h_texturesCfg.imageTextureCount, &d_imageTextures, d_texturesCfg, &(d_texturesCfg->imageTextures));
    }

    // Copy the scalar values from host to device for gradient color texture count
    hipMemcpy(&(d_texturesCfg->imageTextureCount), &(h_texturesCfg.imageTextureCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Checker textures
    if (h_texturesCfg.checkerTextureCount > 0)
    {
        checkerTextureConfig* d_checkerTextures;
        copyCheckerTextureConfig(h_texturesCfg.checkerTextures, h_texturesCfg.checkerTextureCount, &d_checkerTextures, d_texturesCfg, &(d_texturesCfg->checkerTextures));
    }

    // Copy the scalar values from host to device for gradient color texture count
    hipMemcpy(&(d_texturesCfg->checkerTextureCount), &(h_texturesCfg.checkerTextureCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Noise textures
    if (h_texturesCfg.noiseTextureCount > 0)
    {
        noiseTextureConfig* d_noiseTextures;
        copyCommonTextureConfig(h_texturesCfg.noiseTextures, h_texturesCfg.noiseTextureCount, &d_noiseTextures, d_texturesCfg, &(d_texturesCfg->noiseTextures));
    }

    // Copy the scalar values from host to device for gradient color texture count
    hipMemcpy(&(d_texturesCfg->noiseTextureCount), &(h_texturesCfg.noiseTextureCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Bump textures
    if (h_texturesCfg.bumpTextureCount > 0)
    {
        bumpTextureConfig* d_bumpTextures;
        copyImageTextureConfig(h_texturesCfg.bumpTextures, h_texturesCfg.bumpTextureCount, &d_bumpTextures, d_texturesCfg, &(d_texturesCfg->bumpTextures));
    }

    // Copy the scalar values from host to device for gradient color texture count
    hipMemcpy(&(d_texturesCfg->bumpTextureCount), &(h_texturesCfg.bumpTextureCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Normal textures
    if (h_texturesCfg.normalTextureCount > 0)
    {
        normalTextureConfig* d_normalTextures;
        copyImageTextureConfig(h_texturesCfg.normalTextures, h_texturesCfg.normalTextureCount, &d_normalTextures, d_texturesCfg, &(d_texturesCfg->normalTextures));
    }

    // Copy the scalar values from host to device for gradient color texture count
    hipMemcpy(&(d_texturesCfg->normalTextureCount), &(h_texturesCfg.normalTextureCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Displacement textures
    if (h_texturesCfg.displacementTextureCount > 0)
    {
        displacementTextureConfig* d_displacementTextures;
        copyImageTextureConfig(h_texturesCfg.displacementTextures, h_texturesCfg.displacementTextureCount, &d_displacementTextures, d_texturesCfg, &(d_texturesCfg->displacementTextures));
    }

    // Copy the scalar values from host to device for gradient color texture count
    hipMemcpy(&(d_texturesCfg->displacementTextureCount), &(h_texturesCfg.displacementTextureCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Alpha textures
    if (h_texturesCfg.alphaTextureCount > 0)
    {
        alphaTextureConfig* d_alphaTextures;
        copyImageTextureConfig(h_texturesCfg.alphaTextures, h_texturesCfg.alphaTextureCount, &d_alphaTextures, d_texturesCfg, &(d_texturesCfg->alphaTextures));
    }

    // Copy the scalar values from host to device for gradient color texture count
    hipMemcpy(&(d_texturesCfg->alphaTextureCount), &(h_texturesCfg.alphaTextureCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Emissive textures
    if (h_texturesCfg.emissiveTextureCount > 0)
    {
        emissiveTextureConfig* d_emissiveTextures;
        copyImageTextureConfig(h_texturesCfg.emissiveTextures, h_texturesCfg.emissiveTextureCount, &d_emissiveTextures, d_texturesCfg, &(d_texturesCfg->emissiveTextures));
    }

    // Copy the scalar values from host to device for gradient color texture count
    hipMemcpy(&(d_texturesCfg->emissiveTextureCount), &(h_texturesCfg.emissiveTextureCount), sizeof(int8_t), hipMemcpyHostToDevice);


    return d_texturesCfg;
}

materialsConfig* prepareMaterials(const materialsConfig& h_materialsCfg)
{
    // Allocate and copy the materials data
    materialsConfig* d_materialsCfg;
    hipMalloc((void**)&d_materialsCfg, sizeof(materialsConfig));


    // Lambertian materials
    if (h_materialsCfg.lambertianMaterialCount > 0)
    {
        lambertianMaterialConfig* d_lambertianMaterials;
        copyTextureMaterialConfig(h_materialsCfg.lambertianMaterials, h_materialsCfg.lambertianMaterialCount, &d_lambertianMaterials, d_materialsCfg, &(d_materialsCfg->lambertianMaterials));
    }

    // Copy the scalar values from host to device for lambertian materials count
    hipMemcpy(&(d_materialsCfg->lambertianMaterialCount), &(h_materialsCfg.lambertianMaterialCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Metal materials
    if (h_materialsCfg.metalMaterialCount > 0)
    {
        metalMaterialConfig* d_metalMaterials;
        copyCommonMaterialConfig(h_materialsCfg.metalMaterials, h_materialsCfg.metalMaterialCount, &d_metalMaterials, d_materialsCfg, &(d_materialsCfg->metalMaterials));
    }

    // Copy the scalar values from host to device for metal materials count
    hipMemcpy(&(d_materialsCfg->metalMaterialCount), &(h_materialsCfg.metalMaterialCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Glass materials
    if (h_materialsCfg.dielectricMaterialCount > 0)
    {
        dielectricMaterialConfig* d_dielectricMaterials;
        copyCommonMaterialConfig(h_materialsCfg.dielectricMaterials, h_materialsCfg.dielectricMaterialCount, &d_dielectricMaterials, d_materialsCfg, &(d_materialsCfg->dielectricMaterials));
    }

    // Copy the scalar values from host to device for dielectric materials count
    hipMemcpy(&(d_materialsCfg->dielectricMaterialCount), &(h_materialsCfg.dielectricMaterialCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Isotropic materials
    if (h_materialsCfg.isotropicMaterialCount > 0)
    {
        isotropicMaterialConfig* d_isotropicMaterials;
        copyTextureMaterialConfig(h_materialsCfg.isotropicMaterials, h_materialsCfg.isotropicMaterialCount, &d_isotropicMaterials, d_materialsCfg, &(d_materialsCfg->isotropicMaterials));
    }

    // Copy the scalar values from host to device for isotropic materials count
    hipMemcpy(&(d_materialsCfg->isotropicMaterialCount), &(h_materialsCfg.isotropicMaterialCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Anisotropic materials
    if (h_materialsCfg.anisotropicMaterialCount > 0)
    {
        anisotropicMaterialConfig* d_anisotropicMaterials;
        copyAnisotropicMaterialConfig(h_materialsCfg.anisotropicMaterials, h_materialsCfg.anisotropicMaterialCount, &d_anisotropicMaterials, d_materialsCfg, &(d_materialsCfg->anisotropicMaterials));
    }

    // Copy the scalar values from host to device for anisotropic materials count
    hipMemcpy(&(d_materialsCfg->anisotropicMaterialCount), &(h_materialsCfg.anisotropicMaterialCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Oren nayar materials
    if (h_materialsCfg.orenNayarMaterialCount > 0)
    {
        orenNayarMaterialConfig* d_orenNayarMaterials;
        copyTextureMaterialConfig(h_materialsCfg.orenNayarMaterials, h_materialsCfg.orenNayarMaterialCount, &d_orenNayarMaterials, d_materialsCfg, &(d_materialsCfg->orenNayarMaterials));
    }

    // Copy the scalar values from host to device for oren nayar materials count
    hipMemcpy(&(d_materialsCfg->orenNayarMaterialCount), &(h_materialsCfg.orenNayarMaterialCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Phong materials
    if (h_materialsCfg.phongMaterialCount > 0)
    {
        phongMaterialConfig* d_phongMaterials;
        copyPhongMaterialConfig(h_materialsCfg.phongMaterials, h_materialsCfg.phongMaterialCount, &d_phongMaterials, d_materialsCfg, &(d_materialsCfg->phongMaterials));
    }

    // Copy the scalar values from host to device for phong materials count
    hipMemcpy(&(d_materialsCfg->phongMaterialCount), &(h_materialsCfg.phongMaterialCount), sizeof(int8_t), hipMemcpyHostToDevice);


    return d_materialsCfg;
}


// Main function to prepare lights
lightsConfig* prepareLights(const lightsConfig& h_lightsCfg)
{
    // Allocate and copy the lights data (for omniLights, dirLights, spotLights)
    lightsConfig* d_lightsCfg;
    hipMalloc((void**)&d_lightsCfg, sizeof(lightsConfig));

    // Copy omniLights array if there are omni lights
    if (h_lightsCfg.omniLightCount > 0)
    {
        omniLightConfig* d_omniLights;
        copyLightConfig(h_lightsCfg.omniLights, h_lightsCfg.omniLightCount, &d_omniLights, d_lightsCfg, &(d_lightsCfg->omniLights));
    }

    // Copy the scalar values (like omniLightCount) from host to device
    hipMemcpy(&(d_lightsCfg->omniLightCount), &(h_lightsCfg.omniLightCount), sizeof(int8_t), hipMemcpyHostToDevice);

    // Copy dirLights array if there are directional lights
    if (h_lightsCfg.dirLightCount > 0)
    {
        directionalLightConfig* d_dirLights;
        copyLightConfig(h_lightsCfg.dirLights, h_lightsCfg.dirLightCount, &d_dirLights, d_lightsCfg, &(d_lightsCfg->dirLights));
    }

    // Copy the scalar values (like dirLightCount) from host to device
    hipMemcpy(&(d_lightsCfg->dirLightCount), &(h_lightsCfg.dirLightCount), sizeof(int8_t), hipMemcpyHostToDevice);

    // Copy spotLights array if there are spot lights
    if (h_lightsCfg.spotLightCount > 0)
    {
        spotLightConfig* d_spotLights;
        copyLightConfig(h_lightsCfg.spotLights, h_lightsCfg.spotLightCount, &d_spotLights, d_lightsCfg, &(d_lightsCfg->spotLights));
    }

    // Copy the scalar values (like spotLightCount) from host to device
    hipMemcpy(&(d_lightsCfg->spotLightCount), &(h_lightsCfg.spotLightCount), sizeof(int8_t), hipMemcpyHostToDevice);

    return d_lightsCfg;
}

// Main function to prepare primitives
primitivesConfig* preparePrimitives(const primitivesConfig& h_primitivesCfg)
{
    primitivesConfig* d_primitivesCfg;
    hipMalloc((void**)&d_primitivesCfg, sizeof(primitivesConfig));


    // Sphere primitives
    if (h_primitivesCfg.spherePrimitiveCount > 0)
    {
        spherePrimitiveConfig* d_spherePrimitives;
        copyCommonPrimitiveConfig(h_primitivesCfg.spherePrimitives, h_primitivesCfg.spherePrimitiveCount, &d_spherePrimitives, d_primitivesCfg, &(d_primitivesCfg->spherePrimitives));
    }

    // Copy the scalar values from host to device for sphere primitives count
    hipMemcpy(&(d_primitivesCfg->spherePrimitiveCount), &(h_primitivesCfg.spherePrimitiveCount), sizeof(int8_t), hipMemcpyHostToDevice);



    // Plane primitives
    if (h_primitivesCfg.planePrimitiveCount > 0)
    {
        planePrimitiveConfig* d_planePrimitives;
        copyCommonPrimitiveConfig(h_primitivesCfg.planePrimitives, h_primitivesCfg.planePrimitiveCount, &d_planePrimitives, d_primitivesCfg, &(d_primitivesCfg->planePrimitives));
    }

    // Copy the scalar values from host to device for plane primitives count
    hipMemcpy(&(d_primitivesCfg->planePrimitiveCount), &(h_primitivesCfg.planePrimitiveCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Quad primitives
    if (h_primitivesCfg.quadPrimitiveCount > 0)
    {
        quadPrimitiveConfig* d_quadPrimitives;
        copyCommonPrimitiveConfig(h_primitivesCfg.quadPrimitives, h_primitivesCfg.quadPrimitiveCount, &d_quadPrimitives, d_primitivesCfg, &(d_primitivesCfg->quadPrimitives));
    }

    // Copy the scalar values from host to device for quad primitives count
    hipMemcpy(&(d_primitivesCfg->quadPrimitiveCount), &(h_primitivesCfg.quadPrimitiveCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Box primitives
    if (h_primitivesCfg.boxPrimitiveCount > 0)
    {
        boxPrimitiveConfig* d_boxPrimitives;
        copyCommonPrimitiveConfig(h_primitivesCfg.boxPrimitives, h_primitivesCfg.boxPrimitiveCount, &d_boxPrimitives, d_primitivesCfg, &(d_primitivesCfg->boxPrimitives));
    }

    // Copy the scalar values from host to device for box primitives count
    hipMemcpy(&(d_primitivesCfg->boxPrimitiveCount), &(h_primitivesCfg.boxPrimitiveCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Cone primitives
    if (h_primitivesCfg.conePrimitiveCount > 0)
    {
        conePrimitiveConfig* d_conePrimitives;
        copyCommonPrimitiveConfig(h_primitivesCfg.conePrimitives, h_primitivesCfg.conePrimitiveCount, &d_conePrimitives, d_primitivesCfg, &(d_primitivesCfg->conePrimitives));
    }

    // Copy the scalar values from host to device for cone primitives count
    hipMemcpy(&(d_primitivesCfg->conePrimitiveCount), &(h_primitivesCfg.conePrimitiveCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Cylinder primitives
    if (h_primitivesCfg.cylinderPrimitiveCount > 0)
    {
        cylinderPrimitiveConfig* d_cylinderPrimitives;
        copyCommonPrimitiveConfig(h_primitivesCfg.cylinderPrimitives, h_primitivesCfg.cylinderPrimitiveCount, &d_cylinderPrimitives, d_primitivesCfg, &(d_primitivesCfg->cylinderPrimitives));
    }

    // Copy the scalar values from host to device for cylinder primitives count
    hipMemcpy(&(d_primitivesCfg->cylinderPrimitiveCount), &(h_primitivesCfg.cylinderPrimitiveCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Disk primitives
    if (h_primitivesCfg.diskPrimitiveCount > 0)
    {
        diskPrimitiveConfig* d_diskPrimitives;
        copyCommonPrimitiveConfig(h_primitivesCfg.diskPrimitives, h_primitivesCfg.diskPrimitiveCount, &d_diskPrimitives, d_primitivesCfg, &(d_primitivesCfg->diskPrimitives));
    }

    // Copy the scalar values from host to device for disk primitives count
    hipMemcpy(&(d_primitivesCfg->diskPrimitiveCount), &(h_primitivesCfg.diskPrimitiveCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Torus primitives
    if (h_primitivesCfg.torusPrimitiveCount > 0)
    {
        torusPrimitiveConfig* d_torusPrimitives;
        copyCommonPrimitiveConfig(h_primitivesCfg.torusPrimitives, h_primitivesCfg.torusPrimitiveCount, &d_torusPrimitives, d_primitivesCfg, &(d_primitivesCfg->torusPrimitives));
    }

    // Copy the scalar values from host to device for torus primitives count
    hipMemcpy(&(d_primitivesCfg->torusPrimitiveCount), &(h_primitivesCfg.torusPrimitiveCount), sizeof(int8_t), hipMemcpyHostToDevice);


    // Volume primitives
    if (h_primitivesCfg.volumePrimitiveCount > 0)
    {
        volumePrimitiveConfig* d_volumePrimitives;
        copyVolumePrimitiveConfig(h_primitivesCfg.volumePrimitives, h_primitivesCfg.volumePrimitiveCount, &d_volumePrimitives, d_primitivesCfg, &(d_primitivesCfg->volumePrimitives));
    }

    // Copy the scalar values from host to device for volume primitives count
    hipMemcpy(&(d_primitivesCfg->volumePrimitiveCount), &(h_primitivesCfg.volumePrimitiveCount), sizeof(int8_t), hipMemcpyHostToDevice);

    return d_primitivesCfg;


}

void renderGPU(const sceneConfig& sceneCfg, const hipDeviceProp_t& prop, int width, int height, int spp, int max_depth, int tx, int ty, const char* filepath)
{
    std::cout << "[INFO] Rendering " << width << "x" << height << " " << spp << " samples > " << filepath << std::endl;

    setupCuda(prop);



    float ratio = (float)height / (float)width;
    int sqrt_spp = static_cast<int>(sqrt(spp));
    

    dim3 single_block(1, 1);
    dim3 single_thread(1, 1);


    //int bytes_per_pixel = 3;
    //int tex_x, tex_y, tex_n;
    //unsigned char *tex_data_host = stbi_load("e:\\uv_mapper_no_numbers.jpg", &tex_x, &tex_y, &tex_n, bytes_per_pixel);
    //if (!tex_data_host) {
    //    std::cerr << "[ERROR] Failed to load texture." << std::endl;
    //    return;
    //}

    //unsigned char *tex_data;
    //checkCudaErrors(hipMallocManaged(&tex_data, tex_x * tex_y * tex_n * sizeof(unsigned char)));
    //checkCudaErrors(hipMemcpy(tex_data, tex_data_host, tex_x * tex_y * tex_n * sizeof(unsigned char), hipMemcpyHostToDevice));

    //image_texture** h_texture;
    //checkCudaErrors(hipMalloc((void **)&h_texture, sizeof(image_texture*)));
    //texture_init<<<single_block, single_thread>>>(tex_data, tex_x, tex_y, tex_n, h_texture);




    // Allocating CUDA memory
    color* d_output;
    checkCudaErrors(hipMallocManaged((void**)&d_output, width * height * sizeof(color)));


    scene* world_device;
    checkCudaErrors(hipMalloc((void**)&world_device, sizeof(scene)));
    //checkCudaErrors(hipMemcpy(world_device, &world, sizeof(scene), hipMemcpyHostToDevice));


    int bytes_per_pixel = 3;

    // Number of images (textures)
    int num_textures = sceneCfg.texturesCfg.imageTextureCount;

    // Allocate space for texture pointers on the device
    bitmap_image** h_textures;
    checkCudaErrors(hipMalloc((void**)&h_textures, num_textures * sizeof(bitmap_image*)));

    // Loop over all images and load them
    for (int i = 0; i < num_textures; ++i)
    {
        int tex_x, tex_y, tex_n;
        imageTextureConfig imageTexture = sceneCfg.texturesCfg.imageTextures[i];
        unsigned char* tex_data_host = stbi_load(imageTexture.filepath, &tex_x, &tex_y, &tex_n, bytes_per_pixel);
        if (!tex_data_host) {
            std::cerr << "[ERROR] Failed to load texture: " << imageTexture.filepath << std::endl;
            return;
        }

        // Allocate managed memory for texture data on device
        unsigned char* tex_data;
        checkCudaErrors(hipMallocManaged(&tex_data, tex_x * tex_y * tex_n * sizeof(unsigned char)));
        checkCudaErrors(hipMemcpy(tex_data, tex_data_host, tex_x * tex_y * tex_n * sizeof(unsigned char), hipMemcpyHostToDevice));

        // Initialize texture on device
        bitmap_image** h_texture;
        checkCudaErrors(hipMalloc((void**)&h_texture, sizeof(bitmap_image*)));
        texture_init<<<single_block, single_thread>>>(tex_data, tex_x, tex_y, tex_n, h_texture);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        // Store the pointer to the current texture in the array
        checkCudaErrors(hipMemcpy(&h_textures[i], h_texture, sizeof(bitmap_image*), hipMemcpyDeviceToDevice));

        // Free the host-side texture after copying to the device
        stbi_image_free(tex_data_host);
    }



    sceneConfig* d_sceneCfg;

    // Allocate memory on the device for the top-level `sceneConfig` struct
    hipMalloc((void**)&d_sceneCfg, sizeof(sceneConfig));


    
    texturesConfig* d_texturesCfg = prepareTextures(sceneCfg.texturesCfg);
    materialsConfig* d_materialsCfg = prepareMaterials(sceneCfg.materialsCfg);
    lightsConfig* d_lightsCfg = prepareLights(sceneCfg.lightsCfg);
    primitivesConfig* d_primitivesCfg = preparePrimitives(sceneCfg.primitivesCfg);



    // Now copy the lightsConfig pointer from host to device sceneConfig
    hipMemcpy(&d_sceneCfg->lightsCfg, d_lightsCfg, sizeof(lightsConfig), hipMemcpyHostToDevice);

    // Now copy the texturesConfig pointer from host to device sceneConfig
    hipMemcpy(&d_sceneCfg->texturesCfg, d_texturesCfg, sizeof(texturesConfig), hipMemcpyHostToDevice);

    // Now copy the materialsConfig pointer from host to device sceneConfig
    hipMemcpy(&d_sceneCfg->materialsCfg, d_materialsCfg, sizeof(materialsConfig), hipMemcpyHostToDevice);

    // Now copy the primitivesConfig pointer from host to device sceneConfig
    hipMemcpy(&d_sceneCfg->primitivesCfg, d_primitivesCfg, sizeof(primitivesConfig), hipMemcpyHostToDevice);





    // World
    hittable_list **elist;
    checkCudaErrors(hipMalloc((void**)&elist, sizeof(hittable_list*)));

    hittable_list **elights;
    checkCudaErrors(hipMalloc((void**)&elights, sizeof(hittable_list*)));
    
    camera** cam;
    checkCudaErrors(hipMalloc((void**)&cam, sizeof(camera*)));

    sampler** aa_sampler;
    checkCudaErrors(hipMalloc((void**)&aa_sampler, sizeof(sampler*)));






    load_scene<<<single_block, single_thread>>>(d_sceneCfg, elist, elights, cam, aa_sampler, width, height, ratio, spp, sqrt_spp, h_textures, num_textures, 1984);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    dim3 render_blocks(width / tx+1, height / ty+1);
    dim3 render_threads(tx, ty);


    printf("[INFO] Render with %u/%u blocks of %u/%u threads\n", render_blocks.x, render_blocks.y, render_threads.x, render_threads.y);


    render<<<render_blocks, render_threads>>>(world_device, d_output, width, height, spp, sqrt_spp, max_depth, elist, elights, cam, aa_sampler, 2580);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());


    // little padding to avoid remaining black zone at the end of the render preview
    for (int jj = 0; jj < 4; jj++)
    {
        for (int ii = 0; ii < width; ii++)
        {
            printf("p %u %u %u %u %u\n", ii, height - jj - 1, 0, 0, 0);
        }
    }
    


    uint8_t* imageHost = new uint8_t[width * height * 3 * sizeof(uint8_t)];
    for (int j = height - 1; j >= 0; j--) {
        for (int i = 0; i < width; i++) {
            size_t pixel_index = j * width + i;

            imageHost[(height - j - 1) * width * 3 + i * 3] = (uint8_t)d_output[pixel_index].r();
            imageHost[(height - j - 1) * width * 3 + i * 3 + 1] = (uint8_t)d_output[pixel_index].g();
            imageHost[(height - j - 1) * width * 3 + i * 3 + 2] = (uint8_t)d_output[pixel_index].b();
        }
    }

    // save image
    stbi_write_png(filepath, width, height, 3, imageHost, width * 3);

    // Clean up
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    
    checkCudaErrors(hipFree(cam));
    checkCudaErrors(hipFree(elights));
    checkCudaErrors(hipFree(elist));
    checkCudaErrors(hipFree(world_device));
    checkCudaErrors(hipFree(aa_sampler));
    checkCudaErrors(hipFree(d_output));
    checkCudaErrors(hipFree(d_sceneCfg));
}


void launchGPU(const sceneConfig& sceneCfg, int width, int height, int spp, int max_depth, int tx, int ty, const char* filepath, bool quietMode)
{
    hipDeviceProp_t prop;

    if (!isGpuAvailable(prop))
    {
        return;
    }

    //std::cout << "Rendering222 " << nx << "x" << ny << " " << ns << " samples > " << filepath << std::endl;

    //std::cout << "[INFO] Use GPU device " << deviceIndex << " " << deviceName << std::endl;

    // https://developer.nvidia.com/blog/accelerated-ray-tracing-cuda/
    // __global__ - Runs on the GPU, called from the CPU or the GPU*. Executed with <<<dim3>>> arguments.
    // __device__ - Runs on the GPU, called from the GPU. Can be used with variabiles too.
    // __host__ - Runs on the CPU, called from the CPU.
    // 
    // --expt-relaxed-constexpr -Xcudafe --diag_suppress=esa_on_defaulted_function_ignored --std c++20 --verbose
    // --expt-relaxed-constexpr --std c++20 -Xcudafe="--diag_suppress=20012 --diag_suppress=20208" 
    //
    renderGPU(sceneCfg, prop, width, height, spp, max_depth, tx, ty, filepath);
}


//int main(int argc, char* argv[])
//{
//    launchGPU(256, 144, 10, 2, 16, 16, "e:\\ttt2.png", true);
//}