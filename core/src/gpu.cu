#include "hip/hip_runtime.h"
#include <iostream>
//#include <string>

#include "hip/hip_runtime.h"
#include ""

//#include <stdio.h>

#include <hiprand/hiprand_kernel.h>

#include "misc/vector3.cuh"
#include "misc/bvh_node.cuh"
#include "cameras/camera.cuh"
#include "cameras/perspective_camera.cuh"
#include "primitives/hittable_list.cuh"
#include "primitives/sphere.cuh"
#include "primitives/quad.cuh"
#include "primitives/aarect.cuh"
#include "materials/diffuse_light.cuh"
#include "primitives/moving_sphere.cuh"
#include "materials/lambertian.cuh"
#include "materials/metal.cuh"
#include "materials/dielectric.cuh"
#include "textures/texture.cuh"
#include "textures/solid_color_texture.cuh"
#include "textures/checker_texture.cuh"
#include "textures/image_texture.cuh"
#include "primitives/box.cuh"

#include "materials/isotropic.cuh"
#include "primitives/volume.cuh"

#include "primitives/translate.cuh"
#include "primitives/rotate.cuh"
#include "primitives/scale.cuh"
#include "primitives/flip_normals.cuh"

#include "lights/light.cuh"
#include "lights/omni_light.cuh"
#include "lights/directional_light.cuh"

#include "utilities/bitmap_image.cuh"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb/stb_image.h>
#include <stb/stb_image_write.h>




// https://github.com/Belval/raytracing

bool isGpuAvailable()
{
    int devicesCount;
    hipGetDeviceCount(&devicesCount);
    for (int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
    {
        hipDeviceProp_t deviceProperties;
        hipGetDeviceProperties(&deviceProperties, deviceIndex);
        if (deviceProperties.major >= 2 && deviceProperties.minor >= 0)
        {
            std::cout << "Use GPU device " << deviceIndex << " " << deviceProperties.name << std::endl;
            
            hipSetDevice(deviceIndex);

            return true;
        }
    }

    std::cout << "Use Nvidia Cuda GPU device found" << std::endl;
    return false;
}


#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' " << hipGetErrorString(result) << "\n";
        // reset device before terminating
        hipDeviceReset();
        exit(99);
    }
}

//__device__ color get_color(const ray& r, const color& background, hittable **world, hittable_list* lights, hiprandState *local_rand_state)
//{
//    ray cur_ray = r;
//    color cur_attenuation = color(1.0, 1.0, 1.0);
//    color cur_emitted = color(0.0, 0.0, 0.0);
//
//    //for(int i = 0; i < 100; i++) {
//        hit_record rec;
//        if ((*world)->hit(cur_ray, interval(0.001f, FLT_MAX), rec, 0, local_rand_state))
//        {
//            scatter_record srec;
//            color attenuation;
//            color emitted = rec.mat->emitted(cur_ray, rec, rec.u, rec.v, rec.hit_point, local_rand_state);
//
//            if(rec.mat->scatter(cur_ray, lights, rec, srec, local_rand_state))
//            {
//                cur_attenuation *= attenuation;
//                cur_emitted += emitted * cur_attenuation;
//                //cur_ray = scattered; // ?????????????
//                cur_ray = srec.skip_pdf_ray; // ?????????????
//            }
//            else {
//                return cur_emitted + emitted * cur_attenuation;
//            }
//        }
//        else {
//            return cur_emitted;
//        }
//    //}
//    return cur_emitted; // exceeded recursion
//}

__device__ color ray_color(const ray& r, int i, int j, int depth, hittable_list& _world, hittable_list& _lights, hiprandState* local_rand_state)
{
    // If we've exceeded the ray bounce limit, no more light is gathered.
    if (depth <= 0)
    {
        return color::black();// background_color;
    }

    hit_record rec;

    vector3 unit_dir = unit_vector(r.direction());

    // If the ray hits nothing, return the background color.
    // 0.001 is to fix shadow acne interval
    if (!_world.hit(r, interval(SHADOW_ACNE_FIX, FLT_MAX), rec, depth, local_rand_state))
    {
        //if (background_texture)
        //{
        //    return get_background_image_color(r.x, r.y, unit_dir, background_texture, background_iskybox);
        //}
        //else
        //{
        //    return background_color;
        //}

        return color::black();
    }

    // ray hit a world object
    scatter_record srec;
    color color_from_emission = rec.mat->emitted(r, rec, rec.u, rec.v, rec.hit_point, local_rand_state);

    // hack for invisible primitives (such as lights)
    if (color_from_emission.a() == 0.0f)
    {
        // rethrow a new ray
        _world.hit(r, interval(rec.t + 0.001f, FLT_MAX), rec, depth, local_rand_state);
    }

    if (!rec.mat->scatter(r, _lights, rec, srec, local_rand_state))
    {
        return color_from_emission;
    }

    

    if (_lights.object_count == 0)
    {
        // no lights
        // no importance sampling
        return srec.attenuation * ray_color(srec.skip_pdf_ray, i, j, depth - 1, _world, _lights, local_rand_state);
    }

    // no importance sampling
    if (srec.skip_pdf)
    {
        return srec.attenuation * ray_color(srec.skip_pdf_ray, i, j, depth - 1, _world, _lights, local_rand_state);
    }

    hittable_pdf* hpdf = new hittable_pdf(_lights, rec.hit_point);


    mixture_pdf* mpdf;

    //if (background_texture && background_iskybox)
    //{
    //    mixture_pdf p_objs(light_ptr, srec.pdf_ptr, 0.5f);
    //    p = mixture_pdf(new mixture_pdf(p_objs), background_pdf, 0.8f);
    //}
    //else
    //{
    mpdf = new mixture_pdf(hpdf, srec.pdf_ptr);
    //}


    ray scattered = ray(rec.hit_point, mpdf->generate(srec, local_rand_state), r.time());
    float pdf_val = mpdf->value(scattered.direction(), local_rand_state);
    float scattering_pdf = rec.mat->scattering_pdf(r, rec, scattered);

    color final_color(0,0,0);

    //if (background_texture)
    //{
        // with background image
        //bool double_sided = false;
        //if (rec.mat->has_alpha_texture(double_sided))
        //{
        //    // render transparent object (having an alpha texture)
        //    color background_behind = rec.mat->get_diffuse_pixel_color(rec);

        //    ray ray_behind(rec.hit_point, r.direction(), r.x, r.y, r.time());
        //    color background_infrontof = ray_color(ray_behind, depth - 1, _scene, local_rand_state);

        //    hit_record rec_behind;
        //    if (_scene.get_world().hit(ray_behind, interval(0.001f, INFINITY), rec_behind, depth, local_rand_state))
        //    {
        //        // another object is behind the alpha textured object, display it behind
        //        scatter_record srec_behind;

        //        if (double_sided)
        //        {
        //            if (rec_behind.mat->scatter(ray_behind, _scene.get_emissive_objects(), rec_behind, srec_behind, local_rand_state))
        //            {
        //                final_color = color::blend_colors(background_behind, background_infrontof, srec.alpha_value);
        //            }
        //        }
        //        else
        //        {
        //            if (rec_behind.mat->scatter(ray_behind, _scene.get_emissive_objects(), rec_behind, srec_behind, local_rand_state) && rec.front_face)
        //            {
        //                final_color = color::blend_colors(background_behind, background_infrontof, srec.alpha_value);
        //            }
        //            else
        //            {
        //                final_color = background_infrontof;
        //            }
        //        }
        //    }
        //    else
        //    {
        //        // no other object behind the alpha textured object, just display background image
        //        if (double_sided)
        //        {
        //            final_color = color::blend_colors(color_from_emission + background_behind, ray_color(ray(rec.hit_point, r.direction(), r.x, r.y, r.time()), depth - 1, _scene, local_rand_state), srec.alpha_value);
        //        }
        //        else
        //        {
        //            final_color = get_background_image_color(r.x, r.y, unit_dir, background_texture, background_iskybox);
        //        }
        //    }
        //}
        //else
        //{
        //    // render opaque object
        //    color color_from_scatter = ray_color(scattered, depth - 1, _scene, local_rand_state) / pdf_val;
        //    final_color = color_from_emission + srec.attenuation * scattering_pdf * color_from_scatter;
        //}
    //}
    //else
    //{
        // with background color
        //if (r.x == 100 && r.y == 100)
        //    printf("recurse 3 %i %i %i\n", r.x, r.y, depth - 1);

        color sample_color = ray_color(scattered, i, j, depth - 1, _world, _lights, local_rand_state);
        color color_from_scatter = (srec.attenuation * scattering_pdf * sample_color) / pdf_val;

        //bool double_sided = false;
        //if (rec.mat->has_alpha_texture(double_sided))
        //{
        //    // render transparent object (having an alpha texture)
        //    final_color = color::blend_colors(color_from_emission + color_from_scatter, ray_color(ray(rec.hit_point, r.direction(), r.x, r.y, r.time()), depth - 1, _world, _lights, local_rand_state), srec.alpha_value);
        //}
        //else
        //{
            // render opaque object
            final_color = color_from_emission + color_from_scatter;
        //}
    //}


    delete(mpdf);
    delete(hpdf);


    return final_color;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_cornell_box(hittable_list **elist, hittable_list **elights,  camera **cam, int width, int height, int spp, int sqrt_spp, image_texture** texture, hiprandState *rand_state)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        hiprandState local_rand_state = *rand_state;

        //*myscene = new scene();

        *elights = new hittable_list();

        *elist = new hittable_list();

        //int i = 0;
        (*elist)->add(new rt::flip_normals(new yz_rect(0, 555, 0, 555, 555, new lambertian(new solid_color_texture(color(0.12, 0.45, 0.15))), "MyLeft")));
        (*elist)->add(new yz_rect(0, 555, 0, 555, 0, new lambertian(new solid_color_texture(color(0.65, 0.05, 0.05))), "MyRight"));
        

        (*elist)->add(new xz_rect(0, 555, 0, 555, 0, new lambertian(new solid_color_texture(color(0.73, 0.73, 0.73))), "MyGround"));
        (*elist)->add(new rt::flip_normals(new xz_rect(0, 555, 0, 555, 555, new lambertian(new solid_color_texture(color(0.73, 0.73, 0.73))), "MyTop")));
        (*elist)->add(new rt::flip_normals(new xz_rect(0, 555, 0, 555, 555, new lambertian(new solid_color_texture(color(0.73, 0.73, 0.73))), "MyBottom")));
        
        // back
        (*elist)->add(new quad(point3(0,0,555), vector3(555,0,0), vector3(0,555,0), new lambertian(new solid_color_texture(color(0.73, 0.73, 0.73))), "MyBack"));


        // box
        (*elist)->add(new rt::translate(new box(vector3(0, 0, 295), vector3(165, 330, 165), new lambertian(new solid_color_texture(color(0.73, 0.73, 0.73))), "MyBox"), vector3(120,0,320)));
        

        // sphere
        (*elist)->add(new sphere(vector3(350.0f, 50.0f, 295.0f), 100.0f, new lambertian(*texture), "MySphere"));


        (*elist)->add(new directional_light(point3(278, 554, 332), vector3(-180, 0, 0), vector3(0, 0, -180), 1.5f, color(15, 15, 15), "MyLight", false));




        // temp extract_emissive_objects
        for (int i = 0; i < (*elist)->object_count; i++)
        {
            if ((*elist)->objects[i]->getTypeID() == HittableTypeID::lightDirectionalType)
            {
                light* derived = static_cast<light*>((*elist)->objects[i]);
                if (derived)
                {
                    (*elights)->add((*elist)->objects[i]);
                }
            }
        }

        *cam = new perspective_camera();
        (*cam)->initialize(
            vector3(278, 278, -800),
            vector3(278, 278, 0),
            vector3(0, 1, 0),
            256,
            1.0f,
            40.0f,
            100.0f,
            0.0f,
            1.0f,
            sqrt_spp);

        //printf("test %i/%i\n", (*elist)->object_count, (*elist)->object_capacity);

        //for (int i = 0; i < (*elist)->object_count; i++)
        //    printf("test obj %i %s\n", (*elist)->objects[i]->getTypeID(), (*elist)->objects[i]->getName());

        
        //(*myscene)->set(*elist);
        //(*myscene)->set_camera(*cam);
        //(*myscene)->extract_emissive_objects();
        //(*myscene)->build_optimized_world(local_rand_state);
    }
}

__global__ void rand_init(hiprandState *rand_state)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int maxx, int maxy, hiprandState *rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= maxx) || (j >= maxy)) return;
    int pixel_index = j*maxx + i;
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void texture_init(unsigned char* tex_data, int width, int height, int channels, image_texture** tex)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *tex = new image_texture(bitmap_image(tex_data, width, height, channels));
    }
}

__global__ void render(color* fb, int width, int height, int spp, int sqrt_spp, int max_depth, hittable_list **world, hittable_list **lights, camera** cam, hiprandState *randState)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= width) || (j >= height)) return;

    int pixel_index = j* width + i;
    hiprandState local_rand_state = randState[pixel_index];
    color pixel_color(0, 0, 0);

    // new
    for (int s_j = 0; s_j < sqrt_spp; ++s_j)
    {
        for (int s_i = 0; s_i < sqrt_spp; ++s_i)
        {
            /*float u = float(i + hiprand_uniform(&local_rand_state)) / float(width);
            float v = float(j + hiprand_uniform(&local_rand_state)) / float(height);*/

            ray r = (*cam)->get_ray(i, j, s_i, s_j, nullptr, &local_rand_state);

            // pixel color is progressively being refined
            pixel_color += ray_color(r, i, j, max_depth, **world, **lights, &local_rand_state);
        }
    }

    // old
    //for(int s=0; s < spp; s++)
    //{
    //    float u = float(i + hiprand_uniform(&local_rand_state)) / float(width);
    //    float v = float(j + hiprand_uniform(&local_rand_state)) / float(height);
    //    ray r = (*cam)->get_ray(u, v, &local_rand_state);
    //    pixel_color += get_color(r, background, world, lights, &local_rand_state);
    //}

    randState[pixel_index] = local_rand_state;
    fb[pixel_index] = pixel_color;
}

void renderGPU(int width, int height, int spp, int max_depth, int tx, int ty, const char* filepath)
{
    std::cout << "Rendering " << width << "x" << height << " " << spp << " samples > " << filepath << std::endl;

    size_t stackSize;

    // Get the current stack size limit
    hipError_t result1 = hipDeviceGetLimit(&stackSize, hipLimitStackSize);
    if (result1 != hipSuccess) {
        std::cerr << "Failed to get stack size: " << hipGetErrorString(result1) << std::endl;
        return;
    }

    std::cout << "Current stack size limit: " << stackSize << " bytes" << std::endl;


    const size_t newStackSize = 4096 * 10; // Set the stack size to 1MB per thread

    hipError_t result2 = hipDeviceSetLimit(hipLimitStackSize, newStackSize);
    if (result2 != hipSuccess) {
        std::cerr << "Failed to set stack size: " << hipGetErrorString(result2) << std::endl;
        return;
    }

    std::cout << "New stack size limit: " << newStackSize << " bytes" << std::endl;



    const size_t newMallowHeapSize = size_t(1024) * size_t(1024) * size_t(1024);

    hipError_t result3 = hipDeviceSetLimit(hipLimitMallocHeapSize, newMallowHeapSize);
    if (result3 != hipSuccess) {
        std::cerr << "Failed to set malloc heap size: " << hipGetErrorString(result3) << std::endl;
        return;
    }

    std::cout << "New malloc heap limit: " << newMallowHeapSize << " bytes" << std::endl;


    // cuda initialization via hipMalloc
    //size_t limit = 0;

    //hipDeviceGetLimit(&limit, hipLimitStackSize);
    //printf("hipLimitStackSize: %u\n", (unsigned)limit);
    //hipDeviceGetLimit(&limit, hipLimitPrintfFifoSize);
    //printf("hipLimitPrintfFifoSize: %u\n", (unsigned)limit);
    //hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
    //printf("hipLimitMallocHeapSize: %u\n", (unsigned)limit);

    //std::cout << "default settings of cuda context" << std::endl;
    //
    //limit = 10;

    //hipDeviceSetLimit(hipLimitStackSize, limit);
    //hipDeviceSetLimit(hipLimitPrintfFifoSize, limit);
    //hipDeviceSetLimit(hipLimitMallocHeapSize, limit);

    //std::cout << "set limit to 10 for all settings" << std::endl;
    //

    //limit = 0;

    //hipDeviceGetLimit(&limit, hipLimitStackSize);
    //printf("New hipLimitStackSize: %u\n", (unsigned)limit);
    //hipDeviceGetLimit(&limit, hipLimitPrintfFifoSize);
    //printf("New hipLimitPrintfFifoSize: %u\n", (unsigned)limit);
    //hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
    //printf("New hipLimitMallocHeapSize: %u\n", (unsigned)limit);






    int sqrt_spp = static_cast<int>(sqrt(spp));
    
    // Values
    int num_pixels = width * height;

    int tex_x, tex_y, tex_n;
    unsigned char *tex_data_host = stbi_load("e:\\earth_diffuse.jpg", &tex_x, &tex_y, &tex_n, 0);
    if (!tex_data_host) {
        std::cerr << "Failed to load texture." << std::endl;
        return;
    }

    unsigned char *tex_data;
    checkCudaErrors(hipMallocManaged(&tex_data, tex_x * tex_y * tex_n * sizeof(unsigned char)));
    checkCudaErrors(hipMemcpy(tex_data, tex_data_host, tex_x * tex_y * tex_n * sizeof(unsigned char), hipMemcpyHostToDevice));

    image_texture**texture;
    checkCudaErrors(hipMalloc((void **)&texture, sizeof(image_texture*)));
    texture_init<<<1, 1>>>(tex_data, tex_x, tex_y, tex_n, texture);





    // Allocating CUDA memory
    color* image;
    checkCudaErrors(hipMallocManaged((void**)&image, width * height * sizeof(color)));

    // Allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels * sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1 * sizeof(hiprandState)));

    // Allocate 2nd random state to be initialized for the world creation
    rand_init<<<1,1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Building the world
    hittable_list **elist;
    //int num_entity = 22 * 22 + 1 + 3;
    checkCudaErrors(hipMalloc((void**)&elist, sizeof(hittable_list*)));

    //hittable_list **eworld;
    //checkCudaErrors(hipMalloc((void**)&eworld, sizeof(hittable_list*)));

    hittable_list **elights;
    checkCudaErrors(hipMalloc((void**)&elights, sizeof(hittable_list*)));
    
    camera** cam;
    checkCudaErrors(hipMalloc((void**)&cam, sizeof(camera*)));

    //scene** myscene;
    //checkCudaErrors(hipMalloc((void**)&myscene, sizeof(scene*)));


    create_cornell_box<<<1, 1>>>(elist, elights, cam, width, height, spp, sqrt_spp, texture, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());



    dim3 blocks(width / tx+1, height / ty+1);
    dim3 threads(tx, ty);

    render_init<<<blocks, threads>>>(width, height, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(image, width, height, spp, sqrt_spp, max_depth, elist, elights, cam, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    static const interval intensity(0.000, 0.999);

    uint8_t* imageHost = new uint8_t[width * height * 3 * sizeof(uint8_t)];
    for (int j = 0; j < height; j++) {
        for (int i = 0; i < width; i++) {
            size_t pixel_index = j * width + i;

            color fix = color::prepare_pixel_color(i, j, image[pixel_index], spp, false);

            imageHost[j * width * 3 + i * 3] = 256 * intensity.clamp(fix.r());
            imageHost[j * width * 3 + i * 3 + 1] = 256 * intensity.clamp(fix.g());
            imageHost[j * width * 3 + i * 3 + 2] = 256 * intensity.clamp(fix.b());
        }
    }

    stbi_write_png(filepath, width, height, 3, imageHost, width * 3);

    // Clean up
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(cam));
    checkCudaErrors(hipFree(elights));
    checkCudaErrors(hipFree(elist));
    //checkCudaErrors(hipFree(myscene));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(image));
}


void launchGPU(int width, int height, int spp, int max_depth, int tx, int ty, const char* filepath, bool quietMode)
{
    if (!isGpuAvailable())
    {
        return;
    }

    //std::cout << "Rendering222 " << nx << "x" << ny << " " << ns << " samples > " << filepath << std::endl;

    //std::cout << "[INFO] Use GPU device " << deviceIndex << " " << deviceName << std::endl;

    // https://developer.nvidia.com/blog/accelerated-ray-tracing-cuda/
    // __global__ - Runs on the GPU, called from the CPU or the GPU*. Executed with <<<dim3>>> arguments.
    // __device__ - Runs on the GPU, called from the GPU. Can be used with variabiles too.
    // __host__ - Runs on the CPU, called from the CPU.
    // 
    // --expt-relaxed-constexpr -Xcudafe --diag_suppress=esa_on_defaulted_function_ignored --std c++20 --verbose
    // --expt-relaxed-constexpr --std c++20 -Xcudafe="--diag_suppress=20012 --diag_suppress=20208" 
    //
    renderGPU(width, height, spp, max_depth, tx, ty, filepath);
}


//int main(int argc, char* argv[])
//{
//    launchGPU(256, 144, 10, 2, 16, 16, "e:\\ttt2.png", true);
//}