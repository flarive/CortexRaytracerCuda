#include "hip/hip_runtime.h"
#include <iostream>
//#include <string>

#include "hip/hip_runtime.h"
#include ""

//#include <stdio.h>

#include <hiprand/hiprand_kernel.h>

#include "misc/vector3.cuh"
#include "misc/bvh_node.cuh"

#include "primitives/hittable_list.cuh"

#include "textures/texture.cuh"
#include "textures/solid_color_texture.cuh"
#include "textures/checker_texture.cuh"
#include "textures/image_texture.cuh"
#include "textures/bump_texture.cuh"
#include "textures/normal_texture.cuh"
#include "textures/alpha_texture.cuh"
#include "textures/emissive_texture.cuh"




#include "materials/diffuse_light.cuh"
#include "materials/diffuse_spot_light.cuh"
#include "materials/lambertian.cuh"
#include "materials/metal.cuh"
#include "materials/dielectric.cuh"
#include "materials/isotropic.cuh"



#include "primitives/aarect.cuh"
#include "primitives/box.cuh"
#include "primitives/sphere.cuh"
#include "primitives/quad.cuh"
#include "primitives/volume.cuh"
#include "primitives/torus.cuh"
#include "primitives/cylinder.cuh"
#include "primitives/cone.cuh"

#include "primitives/translate.cuh"
#include "primitives/rotate.cuh"
#include "primitives/scale.cuh"
#include "primitives/flip_normals.cuh"


#include "lights/light.cuh"
#include "lights/omni_light.cuh"
#include "lights/directional_light.cuh"
#include "lights/spot_light.cuh"

#include "cameras/camera.cuh"
#include "cameras/perspective_camera.cuh"
#include "cameras/orthographic_camera.cuh"

#include "samplers/sampler.cuh"
#include "samplers/random_sampler.cuh"

#include "utilities/bitmap_image.cuh"


#include "scenes/scene_loader.h"
#include "scenes/scene_builder.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb/stb_image.h>
#include <stb/stb_image_write.h>




// https://github.com/Belval/raytracing

bool isGpuAvailable(hipDeviceProp_t& prop)
{
    int devicesCount;
    hipGetDeviceCount(&devicesCount);
    for (int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
    {
        hipGetDeviceProperties(&prop, deviceIndex);
        if (prop.major >= 2 && prop.minor >= 0)
        {
            printf("[INFO] Use GPU device %d %s\n", deviceIndex, prop.name);
            printf("[INFO] Number of multiprocessors on device: %d\n", prop.multiProcessorCount);
            printf("[INFO] Total global memory (Gbytes) %.1f\n", (float)(prop.totalGlobalMem) / 1024.0 / 1024.0 / 1024.0);
            printf("[INFO] Max grid size: %i x %i x %i\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
            printf("[INFO] Max block size: %i x %i x %i\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
            printf("[INFO] Max number of threads per block: %i\n", prop.maxThreadsPerBlock);

            hipSetDevice(deviceIndex);

            return true;
        }
    }

    std::cout << "[ERROR] No Nvidia Cuda GPU device found" << std::endl;
    return false;
}


#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' " << hipGetErrorString(result) << "\n";
        // reset device before terminating
        hipDeviceReset();
        exit(99);
    }
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void load_scene(hittable_list **elist, hittable_list **elights,  camera **cam, sampler **aa_sampler, int width, int height, float ratio, int spp, int sqrt_spp, image_texture** texture, hiprandState *rand_state)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        hiprandState local_rand_state = *rand_state;

        //*myscene = new scene();

        *elights = new hittable_list();

        *elist = new hittable_list();

        

        (*elist)->add(new rt::flip_normals(new yz_rect(0, 555, 0, 555, 555, new lambertian(new solid_color_texture(color(0.12, 0.45, 0.15))), "MyLeft")));
        (*elist)->add(new yz_rect(0, 555, 0, 555, 0, new lambertian(new solid_color_texture(color(0.65, 0.05, 0.05))), "MyRight"));
        (*elist)->add(new xz_rect(0, 555, 0, 555, 0, new lambertian(new solid_color_texture(color(0.73, 0.73, 0.73))), "MyGround"));
        (*elist)->add(new rt::flip_normals(new xz_rect(0, 555, 0, 555, 555, new lambertian(new solid_color_texture(color(0.73, 0.73, 0.73))), "MyTop")));
        (*elist)->add(new rt::flip_normals(new xz_rect(0, 555, 0, 555, 555, new lambertian(new solid_color_texture(color(0.73, 0.73, 0.73))), "MyBottom")));
        
        // back
        (*elist)->add(new quad(point3(0,0,555), vector3(555,0,0), vector3(0,555,0), new lambertian(new solid_color_texture(color(0.73, 0.73, 0.73))), "MyBack"));


        // box
        (*elist)->add(new rt::translate(new box(vector3(0, 0, 295), vector3(165, 330, 165), new lambertian(*texture), "MyBox"), vector3(120,0,320)));
        
        // sphere
        (*elist)->add(new sphere(vector3(350.0f, 50.0f, 295.0f), 100.0f, new lambertian(*texture), "MySphere"));

        // light
        (*elist)->add(new directional_light(point3(278, 554, 332), vector3(-305, 0, 0), vector3(0, 0, -305), 1.0f, color(10.0, 10.0, 10.0), "MyLight", true));




        // temp extract_emissive_objects
        for (int i = 0; i < (*elist)->object_count; i++)
        {
            if ((*elist)->objects[i]->getTypeID() == HittableTypeID::lightDirectionalType)
            {
                light* derived = static_cast<light*>((*elist)->objects[i]);
                if (derived)
                {
                    (*elights)->add((*elist)->objects[i]);
                }
            }
        }

        *cam = new perspective_camera();
        (*cam)->initialize(
            vector3(278, 278, -800),
            vector3(278, 278, 0),
            vector3(0, 1, 0),
            width,
            ratio,
            40.0f,
            0.0f,
            10.0f,
            0.0f,
            0.0f,
            1.0f,
            sqrt_spp);


        //*aa_sampler = new random_sampler((*cam)->get_pixel_delta_u(), (*cam)->get_pixel_delta_v(), 50);


        // calculate bounding boxes to speed up ray computing
        *elist = new hittable_list(new bvh_node((*elist)->objects, 0, (*elist)->object_count, &local_rand_state));
    }
}

__global__ void rand_init(hiprandState *rand_state)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int maxx, int maxy, hiprandState *rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= maxx) || (j >= maxy)) return;
    int pixel_index = j*maxx + i;
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void texture_init(unsigned char* tex_data, int width, int height, int channels, image_texture** tex)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *tex = new image_texture(bitmap_image(tex_data, width, height, channels));
    }
}

//__global__ void render(color* fb, int width, int height, int spp, int sqrt_spp, int max_depth, hittable_list **world, hittable_list **lights, camera** cam, sampler** aa_sampler, hiprandState *randState)
//{
//    int i = threadIdx.x + blockIdx.x * blockDim.x;
//    int j = threadIdx.y + blockIdx.y * blockDim.y;
//    if ((i >= width) || (j >= height))
//        return;
//    
//        
//
//    int pixel_index = j * width + i;
//    hiprandState local_rand_state = randState[pixel_index];
//    color pixel_color(0, 0, 0);
//
//    for (int s_j = 0; s_j < sqrt_spp; ++s_j)
//    {
//        for (int s_i = 0; s_i < sqrt_spp; ++s_i)
//        {
//            float uniform_random = hiprand_uniform(&local_rand_state);
//            float gaussian_random = hiprand_normal(&local_rand_state);
//
//
//            // Subpixel Sampling for Anti-Aliasing
//            // Normalized Device Coordinates (NDC)
//            // By using u and v, you introduce small random perturbations within each pixel. These perturbations generate different rays that pass through slightly different positions in the scene. This increases the accuracy of the final color calculation for each pixel when averaged over multiple samples (spp or samples per pixel), producing smoother gradients and less noise.
//            float u = float(i + gaussian_random) / float(width);
//            float v = float(j + gaussian_random) / float(height);
//
//            // Stratified sampling with jittered randomness
//            //float u = (i + (s_i + hiprand_uniform(&local_rand_state)) / sqrt_spp) / float(width);
//            //float v = (j + (s_j + hiprand_uniform(&local_rand_state)) / sqrt_spp) / float(height);
//
//            
//
//
//            ray r = (*cam)->get_ray(u, v, s_i, s_j, nullptr, &local_rand_state);
//
//            // pixel color is progressively being refined
//            pixel_color += (*cam)->ray_color(r, i, j, max_depth, **world, **lights, &local_rand_state);
//        }
//    }
//
//    const color& fix = prepare_pixel_color(i, j, pixel_color, spp, true);
//
//    const interval intensity(0.000f, 0.999f);
//
//
//    randState[pixel_index] = local_rand_state;
//
//    int color_r = static_cast<int>(255.99f * intensity.clamp(fix.r()));
//    int color_g = static_cast<int>(255.99f * intensity.clamp(fix.g()));
//    int color_b = static_cast<int>(255.99f * intensity.clamp(fix.b()));
//
//
//    fb[pixel_index] = color(
//        color_r,
//        color_g,
//        color_b
//    );
//
//    printf(
//        "p %u %u %u %u %u\n",
//        i,
//        height - j - 1,
//        color_r,
//        color_g,
//        color_b
//    );
//}


__global__ void render(color* fb, int width, int height, int spp, int sqrt_spp, int max_depth, hittable_list** world, hittable_list** lights, camera** cam, sampler** aa_sampler, hiprandState* randState)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= width) || (j >= height)) return;

    int pixel_index = j * width + i;
    hiprandState local_rand_state = randState[pixel_index];
    color pixel_color(0, 0, 0);

    float uniform_random = hiprand_uniform(&local_rand_state);
    //float normal_random = hiprand_normal(&local_rand_state);
    //int poisson_random = hiprand_poisson(&local_rand_state, 100.0f);

    for (int s_j = 0; s_j < sqrt_spp; ++s_j)
    {
        for (int s_i = 0; s_i < sqrt_spp; ++s_i)
        {
            // Stratified sampling within the pixel, with Sobol randomness
            float u = (i + (s_i + uniform_random) / sqrt_spp) / float(width);
            float v = (j + (s_j + uniform_random) / sqrt_spp) / float(height);

            ray r = (*cam)->get_ray(u, v, s_i, s_j, nullptr, &local_rand_state);
            pixel_color += (*cam)->ray_color(r, i, j, max_depth, max_depth, **world, **lights, &local_rand_state);
        }
    }

    const color& fix = prepare_pixel_color(i, j, pixel_color, spp, true);
    const interval intensity(0.000f, 0.999f);

    randState[pixel_index] = local_rand_state;

    int color_r = static_cast<int>(255.99f * intensity.clamp(fix.r()));
    int color_g = static_cast<int>(255.99f * intensity.clamp(fix.g()));
    int color_b = static_cast<int>(255.99f * intensity.clamp(fix.b()));

    fb[pixel_index] = color(color_r, color_g, color_b);

    printf("p %u %u %u %u %u\n", i, height - j - 1, color_r, color_g, color_b);
}


void setupCuda(const hipDeviceProp_t& prop)
{
    // If you get a null pointer (either from device malloc or device new) you have run out of heap space.
    // https://forums.developer.nvidia.com/t/allocating-memory-from-device-and-cudalimitmallocheapsize/70441
    
    size_t stackSize;

    // Get the current stack size limit
    hipError_t result1 = hipDeviceGetLimit(&stackSize, hipLimitStackSize);
    if (result1 != hipSuccess) {
        std::cerr << "[WARNING] Failed to get stack size: " << hipGetErrorString(result1) << std::endl;
        return;
    }

    std::cout << "[INFO] Current stack size limit: " << stackSize << " bytes" << std::endl;


    const size_t newStackSize = 4096; // Set the stack size to 1MB per thread

    hipError_t result2 = hipDeviceSetLimit(hipLimitStackSize, newStackSize);
    if (result2 != hipSuccess) {
        std::cerr << "[WARNING] Failed to set stack size: " << hipGetErrorString(result2) << std::endl;
        return;
    }

    std::cout << "[INFO] New stack size limit: " << newStackSize << " bytes" << std::endl;



    const size_t newMallocHeapSize = size_t(1024) * size_t(1024) * size_t(1024);

    hipError_t result3 = hipDeviceSetLimit(hipLimitMallocHeapSize, newMallocHeapSize);
    if (result3 != hipSuccess) {
        std::cerr << "[WARNING] Failed to set malloc heap size: " << hipGetErrorString(result3) << std::endl;
        return;
    }

    std::cout << "[INFO] New malloc heap limit: " << newMallocHeapSize << " bytes" << std::endl;



    const size_t newPrintfFifoSize = 10000000;

    hipError_t result4 = hipDeviceSetLimit(hipLimitPrintfFifoSize, newPrintfFifoSize);
    if (result4 != hipSuccess) {
        std::cerr << "[WARNING] Failed to set printf fifo size: " << hipGetErrorString(result4) << std::endl;
        return;
    }

    std::cout << "[INFO] New printf fifo size: " << newPrintfFifoSize << " bytes" << std::endl;
}

void renderGPU(const hipDeviceProp_t& prop, int width, int height, int spp, int max_depth, int tx, int ty, const char* filepath)
{
    std::cout << "[INFO] Rendering " << width << "x" << height << " " << spp << " samples > " << filepath << std::endl;

    setupCuda(prop);



    float ratio = (float)height / (float)width;


    int sqrt_spp = static_cast<int>(sqrt(spp));
    
    // Values
    int num_pixels = width * height;

    int bytes_per_pixel = 3;
    int tex_x, tex_y, tex_n;
    unsigned char *tex_data_host = stbi_load("d:\\uv_mapper_no_numbers.jpg", &tex_x, &tex_y, &tex_n, bytes_per_pixel);
    if (!tex_data_host) {
        std::cerr << "[ERROR] Failed to load texture." << std::endl;
        return;
    }

    unsigned char *tex_data;
    checkCudaErrors(hipMallocManaged(&tex_data, tex_x * tex_y * tex_n * sizeof(unsigned char)));
    checkCudaErrors(hipMemcpy(tex_data, tex_data_host, tex_x * tex_y * tex_n * sizeof(unsigned char), hipMemcpyHostToDevice));



    dim3 single_block(1, 1);
    dim3 single_thread(1, 1);



    image_texture**texture;
    checkCudaErrors(hipMalloc((void **)&texture, sizeof(image_texture*)));
    texture_init<<<single_block, single_thread>>>(tex_data, tex_x, tex_y, tex_n, texture);





    // Allocating CUDA memory
    color* image;
    checkCudaErrors(hipMallocManaged((void**)&image, width * height * sizeof(color)));

    // Allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels * sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1 * sizeof(hiprandState)));

    // Allocate 2nd random state to be initialized for the world creation
    rand_init<<<single_block, single_thread>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Building the world
    hittable_list **elist;
    checkCudaErrors(hipMalloc((void**)&elist, sizeof(hittable_list*)));

    hittable_list **elights;
    checkCudaErrors(hipMalloc((void**)&elights, sizeof(hittable_list*)));
    
    camera** cam;
    checkCudaErrors(hipMalloc((void**)&cam, sizeof(camera*)));

    sampler** aa_sampler;
    checkCudaErrors(hipMalloc((void**)&aa_sampler, sizeof(sampler*)));


    //scene** myscene;
    //checkCudaErrors(hipMalloc((void**)&myscene, sizeof(scene*)));


    load_scene<<<single_block, single_thread>>>(elist, elights, cam, aa_sampler, width, height, ratio, spp, sqrt_spp, texture, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    dim3 render_blocks(width / tx+1, height / ty+1);
    dim3 render_threads(tx, ty);

    render_init<<<render_blocks, render_threads>>>(width, height, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    printf("[INFO] Render with %u/%u blocks of %u/%u threads\n", render_blocks.x, render_blocks.y, render_threads.x, render_threads.y);


    render<<<render_blocks, render_threads>>>(image, width, height, spp, sqrt_spp, max_depth, elist, elights, cam, aa_sampler, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());


    // little padding to avoid remaining black zone at the end of the render preview
    for (int jj = 0; jj < 4; jj++)
    {
        for (int ii = 0; ii < width; ii++)
        {
            printf(
                "p %u %u %u %u %u\n",
                ii,
                height - jj - 1,
                0,
                0,
                0
            );
        }
    }
    


    uint8_t* imageHost = new uint8_t[width * height * 3 * sizeof(uint8_t)];
    for (int j = height - 1; j >= 0; j--) {
        for (int i = 0; i < width; i++) {
            size_t pixel_index = j * width + i;

            imageHost[(height - j - 1) * width * 3 + i * 3] = (size_t)image[pixel_index].r();
            imageHost[(height - j - 1) * width * 3 + i * 3 + 1] = (size_t)image[pixel_index].g();
            imageHost[(height - j - 1) * width * 3 + i * 3 + 2] = (size_t)image[pixel_index].b();
        }
    }

    stbi_write_png(filepath, width, height, 3, imageHost, width * 3);

    // Clean up
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    
    checkCudaErrors(hipFree(cam));
    checkCudaErrors(hipFree(elights));
    checkCudaErrors(hipFree(elist));
    //checkCudaErrors(hipFree(myscene));
    checkCudaErrors(hipFree(aa_sampler));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(image));
}


void launchGPU(int width, int height, int spp, int max_depth, int tx, int ty, const char* filepath, bool quietMode)
{
    hipDeviceProp_t prop;
    
    if (!isGpuAvailable(prop))
    {
        return;
    }

    //std::cout << "Rendering222 " << nx << "x" << ny << " " << ns << " samples > " << filepath << std::endl;

    //std::cout << "[INFO] Use GPU device " << deviceIndex << " " << deviceName << std::endl;

    // https://developer.nvidia.com/blog/accelerated-ray-tracing-cuda/
    // __global__ - Runs on the GPU, called from the CPU or the GPU*. Executed with <<<dim3>>> arguments.
    // __device__ - Runs on the GPU, called from the GPU. Can be used with variabiles too.
    // __host__ - Runs on the CPU, called from the CPU.
    // 
    // --expt-relaxed-constexpr -Xcudafe --diag_suppress=esa_on_defaulted_function_ignored --std c++20 --verbose
    // --expt-relaxed-constexpr --std c++20 -Xcudafe="--diag_suppress=20012 --diag_suppress=20208" 
    //
    renderGPU(prop, width, height, spp, max_depth, tx, ty, filepath);
}


//int main(int argc, char* argv[])
//{
//    launchGPU(256, 144, 10, 2, 16, 16, "e:\\ttt2.png", true);
//}