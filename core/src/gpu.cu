#include "hip/hip_runtime.h"
#include <iostream>




// cuda
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include <thrust/random.h>


#include "misc/vector3.cuh"
#include "misc/bvh_node.cuh"


#include "primitives/hittable_list.cuh"

#include "textures/texture.cuh"
#include "textures/solid_color_texture.cuh"
#include "textures/checker_texture.cuh"
#include "textures/image_texture.cuh"
#include "textures/bump_texture.cuh"
#include "textures/normal_texture.cuh"
#include "textures/alpha_texture.cuh"
#include "textures/emissive_texture.cuh"




#include "materials/diffuse_light.cuh"
#include "materials/diffuse_spot_light.cuh"
#include "materials/lambertian.cuh"
#include "materials/metal.cuh"
#include "materials/dielectric.cuh"
#include "materials/isotropic.cuh"
#include "materials/oren_nayar.cuh"




#include "primitives/aarect.cuh"
#include "primitives/box.cuh"
#include "primitives/sphere.cuh"
#include "primitives/quad.cuh"
#include "primitives/volume.cuh"
#include "primitives/torus.cuh"
#include "primitives/cylinder.cuh"
#include "primitives/cone.cuh"
#include "primitives/disk.cuh"
#include "primitives/triangle.cuh"

#include "primitives/translate.cuh"
#include "primitives/rotate.cuh"
#include "primitives/scale.cuh"
#include "primitives/flip_normals.cuh"


#include "lights/light.cuh"
#include "lights/omni_light.cuh"
#include "lights/directional_light.cuh"
#include "lights/spot_light.cuh"

#include "cameras/camera.cuh"
#include "cameras/perspective_camera.cuh"
#include "cameras/orthographic_camera.cuh"

#include "samplers/sampler.cuh"
#include "samplers/random_sampler.cuh"

#include "utilities/bitmap_image.cuh"


#include "scenes/scene_loader.h"
#include "scenes/scene_builder.h"



#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb/stb_image.h>
#include <stb/stb_image_write.h>




// https://github.com/Belval/raytracing

bool isGpuAvailable(hipDeviceProp_t& prop)
{
    int devicesCount;
    hipGetDeviceCount(&devicesCount);
    for (int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
    {
        hipGetDeviceProperties(&prop, deviceIndex);
        if (prop.major >= 2 && prop.minor >= 0)
        {
            printf("[INFO] Use GPU device %d %s\n", deviceIndex, prop.name);
            printf("[INFO] Number of multiprocessors on device: %d\n", prop.multiProcessorCount);
            printf("[INFO] Total global memory (Gbytes) %.1f\n", (float)(prop.totalGlobalMem) / 1024.0 / 1024.0 / 1024.0);
            printf("[INFO] Max grid size: %i x %i x %i\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
            printf("[INFO] Max block size: %i x %i x %i\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
            printf("[INFO] Max number of threads per block: %i\n", prop.maxThreadsPerBlock);

            hipSetDevice(deviceIndex);

            return true;
        }
    }

    std::cout << "[ERROR] No Nvidia Cuda GPU device found" << std::endl;
    return false;
}


#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' " << hipGetErrorString(result) << "\n";
        // reset device before terminating
        hipDeviceReset();
        exit(99);
    }
}

//#define RND (hiprand_uniform(&rng))

__global__ void load_scene(sceneConfig* sceneCfg, hittable_list **elist, hittable_list **elights,  camera **cam, sampler **aa_sampler, int width, int height, float ratio, int spp, int sqrt_spp, image_texture** texture, int seed)
{
    
    





    
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        // thrust random engine and distribution
        thrust::minstd_rand rng(seed);
        thrust::uniform_real_distribution<float> uniform_dist(0.0f, 1.0f);


        //*myscene = new scene();

        *elights = new hittable_list();

        *elist = new hittable_list();


        //int aa = sceneCfg->lightsCfg.dirLightCount;

        //printf("ZZZZZZZZZZZZ COUNT %i !!!!!!!!!!\n", aa);



        //directionalLightConfig light1 = sceneCfg->lightsCfg.dirLights[0];
        //printf("light1 %g %s !!!!!!!!!!\n", light1.intensity, light1.name);

        //directionalLightConfig light2 = sceneCfg->lightsCfg.dirLights[1];
        //printf("light2 %g %s !!!!!!!!!!\n", light2.intensity, light2.name);





        (*elist)->add(new rt::flip_normals(new yz_rect(0, 555, 0, 555, 555, new lambertian(new solid_color_texture(color(0.12, 0.45, 0.15))), "MyLeft")));
        (*elist)->add(new yz_rect(0, 555, 0, 555, 0, new lambertian(new solid_color_texture(color(0.65, 0.05, 0.05))), "MyRight"));
        (*elist)->add(new xz_rect(0, 555, 0, 555, 0, new lambertian(new solid_color_texture(color(0.73, 0.73, 0.73))), "MyGround"));
        (*elist)->add(new rt::flip_normals(new xz_rect(0, 555, 0, 555, 555, new lambertian(new solid_color_texture(color(0.73, 0.73, 0.73))), "MyTop")));
        (*elist)->add(new rt::flip_normals(new xz_rect(0, 555, 0, 555, 555, new lambertian(new solid_color_texture(color(0.73, 0.73, 0.73))), "MyBottom")));
        
        // back
        (*elist)->add(new quad(point3(0,0,555), vector3(555,0,0), vector3(0,555,0), new lambertian(new solid_color_texture(color(0.73, 0.73, 0.73))), "MyBack"));


        // box
        (*elist)->add(new rt::translate(new box(point3(0.0f, 0.0f, 200.0f), vector3(165, 330, 165), new lambertian(*texture), "MyBox"), vector3(120,0,320)));
        
        // sphere
        (*elist)->add(new sphere(point3(350.0f, 50.0f, 295.0f), 100.0f, new lambertian(*texture), "MySphere"));

        // torus
        //(*elist)->add(new torus(point3(200.0f, 50.0f, 295.0f), 3.0f, 1.0f, new lambertian(*texture), "MyTorus"));

        // light
        (*elist)->add(new directional_light(point3(278, 554, 332), vector3(-305, 0, 0), vector3(0, 0, -305), 1.0f, color(10.0, 10.0, 10.0), "MyLight", true));




        // temp extract_emissive_objects
        for (int i = 0; i < (*elist)->object_count; i++)
        {
            if ((*elist)->objects[i]->getTypeID() == HittableTypeID::lightDirectionalType)
            {
                light* derived = static_cast<light*>((*elist)->objects[i]);
                if (derived)
                {
                    (*elights)->add((*elist)->objects[i]);
                }
            }
        }

        *cam = new perspective_camera();
        (*cam)->initialize(
            vector3(278, 278, -800),
            vector3(278, 278, 0),
            vector3(0, 1, 0),
            width,
            ratio,
            40.0f,
            0.0f,
            10.0f,
            0.0f,
            0.0f,
            1.0f,
            sqrt_spp);


        //*aa_sampler = new random_sampler((*cam)->get_pixel_delta_u(), (*cam)->get_pixel_delta_v(), 50);


        // calculate bounding boxes to speed up ray computing
        *elist = new hittable_list(new bvh_node((*elist)->objects, 0, (*elist)->object_count, rng));
    }
}

__global__ void texture_init(unsigned char* tex_data, int width, int height, int channels, image_texture** tex)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *tex = new image_texture(bitmap_image(tex_data, width, height, channels));
    }
}

__global__ void render(scene* world_scene, color* fb, int width, int height, int spp, int sqrt_spp, int max_depth, hittable_list** world, hittable_list** lights, camera** cam, sampler** aa_sampler, int seed)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= width) || (j >= height)) return;

    int pixel_index = j * width + i;

    // Initialize the random engine and distribution
    thrust::minstd_rand rng(seed + pixel_index);
    thrust::uniform_real_distribution<float> uniform_dist(0.0f, 1.0f);


    color pixel_color(0, 0, 0);

    for (int s_j = 0; s_j < sqrt_spp; ++s_j)
    {
        for (int s_i = 0; s_i < sqrt_spp; ++s_i)
        {
            // Generate a random value between 0 and 1
            float uniform_random = uniform_dist(rng);

            // Stratified sampling within the pixel, with Sobol randomness
            float u = (i + (s_i + uniform_random) / sqrt_spp) / float(width);
            float v = (j + (s_j + uniform_random) / sqrt_spp) / float(height);

            ray r = (*cam)->get_ray(u, v, s_i, s_j, nullptr, rng);
            pixel_color += (*cam)->ray_color(r, i, j, max_depth, max_depth, **world, **lights, rng);
        }
    }

    const color& fix = prepare_pixel_color(i, j, pixel_color, spp, true);
    const interval intensity(0.000f, 0.999f);

    int color_r = static_cast<int>(255.99f * intensity.clamp(fix.r()));
    int color_g = static_cast<int>(255.99f * intensity.clamp(fix.g()));
    int color_b = static_cast<int>(255.99f * intensity.clamp(fix.b()));

    fb[pixel_index] = color(color_r, color_g, color_b);

    printf("p %u %u %u %u %u\n", i, height - j - 1, color_r, color_g, color_b);
}


void setupCuda(const hipDeviceProp_t& prop)
{
    // If you get a null pointer (either from device malloc or device new) you have run out of heap space.
    // https://forums.developer.nvidia.com/t/allocating-memory-from-device-and-cudalimitmallocheapsize/70441
    
    size_t stackSize;

    // Get the current stack size limit
    hipError_t result1 = hipDeviceGetLimit(&stackSize, hipLimitStackSize);
    if (result1 != hipSuccess) {
        std::cerr << "[WARNING] Failed to get stack size: " << hipGetErrorString(result1) << std::endl;
        return;
    }

    std::cout << "[INFO] Current stack size limit: " << stackSize << " bytes" << std::endl;


    const size_t newStackSize = 4096; // Set the stack size to 1MB per thread

    hipError_t result2 = hipDeviceSetLimit(hipLimitStackSize, newStackSize);
    if (result2 != hipSuccess) {
        std::cerr << "[WARNING] Failed to set stack size: " << hipGetErrorString(result2) << std::endl;
        return;
    }

    std::cout << "[INFO] New stack size limit: " << newStackSize << " bytes" << std::endl;



    const size_t newMallocHeapSize = size_t(1024) * size_t(1024) * size_t(1024);

    hipError_t result3 = hipDeviceSetLimit(hipLimitMallocHeapSize, newMallocHeapSize);
    if (result3 != hipSuccess) {
        std::cerr << "[WARNING] Failed to set malloc heap size: " << hipGetErrorString(result3) << std::endl;
        return;
    }

    std::cout << "[INFO] New malloc heap limit: " << newMallocHeapSize << " bytes" << std::endl;



    const size_t newPrintfFifoSize = 10000000;

    hipError_t result4 = hipDeviceSetLimit(hipLimitPrintfFifoSize, newPrintfFifoSize);
    if (result4 != hipSuccess) {
        std::cerr << "[WARNING] Failed to set printf fifo size: " << hipGetErrorString(result4) << std::endl;
        return;
    }

    std::cout << "[INFO] New printf fifo size: " << newPrintfFifoSize << " bytes" << std::endl;
}

void renderGPU(const sceneConfig& sceneCfg, const hipDeviceProp_t& prop, int width, int height, int spp, int max_depth, int tx, int ty, const char* filepath)
{
    std::cout << "[INFO] Rendering " << width << "x" << height << " " << spp << " samples > " << filepath << std::endl;

    setupCuda(prop);



    float ratio = (float)height / (float)width;
    int sqrt_spp = static_cast<int>(sqrt(spp));
    


    int bytes_per_pixel = 3;
    int tex_x, tex_y, tex_n;
    unsigned char *tex_data_host = stbi_load("e:\\uv_mapper_no_numbers.jpg", &tex_x, &tex_y, &tex_n, bytes_per_pixel);
    if (!tex_data_host) {
        std::cerr << "[ERROR] Failed to load texture." << std::endl;
        return;
    }

    unsigned char *tex_data;
    checkCudaErrors(hipMallocManaged(&tex_data, tex_x * tex_y * tex_n * sizeof(unsigned char)));
    checkCudaErrors(hipMemcpy(tex_data, tex_data_host, tex_x * tex_y * tex_n * sizeof(unsigned char), hipMemcpyHostToDevice));



    dim3 single_block(1, 1);
    dim3 single_thread(1, 1);



    image_texture**texture;
    checkCudaErrors(hipMalloc((void **)&texture, sizeof(image_texture*)));
    texture_init<<<single_block, single_thread>>>(tex_data, tex_x, tex_y, tex_n, texture);





    // Allocating CUDA memory
    color* image;
    checkCudaErrors(hipMallocManaged((void**)&image, width * height * sizeof(color)));


    scene* world_device;
    checkCudaErrors(hipMalloc((void**)&world_device, sizeof(scene)));
    //checkCudaErrors(hipMemcpy(world_device, &world, sizeof(scene), hipMemcpyHostToDevice));


    sceneConfig* d_sceneCfg;

    // Allocate memory on the device for the top-level `sceneConfig` struct
    hipMalloc((void**)&d_sceneCfg, sizeof(sceneConfig));

    // Allocate and copy the lights data (for omniLights, dirLights, spotLights)
    lightsConfig* d_lightsCfg;
    hipMalloc((void**)&d_lightsCfg, sizeof(lightsConfig));

    // Copy omniLights
    if (sceneCfg.lightsCfg.omniLightCount > 0) {
        omniLightConfig* d_omniLights;
        hipMalloc((void**)&d_omniLights, sceneCfg.lightsCfg.omniLightCount * sizeof(omniLightConfig));
        hipMemcpy(d_omniLights, sceneCfg.lightsCfg.omniLights, sceneCfg.lightsCfg.omniLightCount * sizeof(omniLightConfig), hipMemcpyHostToDevice);
        // Assign the pointer on device lightsCfg
        hipMemcpy(&d_lightsCfg->omniLights, &d_omniLights, sizeof(omniLightConfig*), hipMemcpyHostToDevice);
    }

    // Copy dirLights array if there are directional lights
    if (sceneCfg.lightsCfg.dirLightCount > 0)
    {
        // 1. Allocate memory for the dirLights array on the device
        directionalLightConfig* d_dirLights;
        hipMalloc((void**)&d_dirLights, sceneCfg.lightsCfg.dirLightCount * sizeof(directionalLightConfig));

        // 2. Copy the dirLights array contents from host to device
        hipMemcpy(d_dirLights, sceneCfg.lightsCfg.dirLights, sceneCfg.lightsCfg.dirLightCount * sizeof(directionalLightConfig), hipMemcpyHostToDevice);

        // 3. Allocate memory and copy the names for each directional light
        for (int i = 0; i < sceneCfg.lightsCfg.dirLightCount; i++)
        {
            const char* hostName = sceneCfg.lightsCfg.dirLights[i].name;  // Get the string from the host

            // Allocate memory on the device for the string (with null terminator)
            char* d_name;
            size_t nameLen = strlen(hostName) + 1;  // +1 for null terminator
            hipMalloc((void**)&d_name, nameLen);

            // Copy the string from host to device
            hipMemcpy(d_name, hostName, nameLen, hipMemcpyHostToDevice);

            // Update the device-side directionalLightConfig to point to the device string
            hipMemcpy(&(d_dirLights[i].name), &d_name, sizeof(char*), hipMemcpyHostToDevice);
        }

        // 4. Update the device-side lightsConfig to point to the dirLights array on the device
        hipMemcpy(&(d_lightsCfg->dirLights), &d_dirLights, sizeof(directionalLightConfig*), hipMemcpyHostToDevice);
    }

    // 5. Copy the scalar values (like dirLightCount) from host to device
    hipMemcpy(&(d_lightsCfg->dirLightCount), &(sceneCfg.lightsCfg.dirLightCount), sizeof(int8_t), hipMemcpyHostToDevice);



    // Copy spotLights
    if (sceneCfg.lightsCfg.spotLightCount > 0) {
        spotLightConfig* d_spotLights;
        hipMalloc((void**)&d_spotLights, sceneCfg.lightsCfg.spotLightCount * sizeof(spotLightConfig));
        hipMemcpy(d_spotLights, sceneCfg.lightsCfg.spotLights, sceneCfg.lightsCfg.spotLightCount * sizeof(spotLightConfig), hipMemcpyHostToDevice);
        hipMemcpy(&d_lightsCfg->spotLights, &d_spotLights, sizeof(spotLightConfig*), hipMemcpyHostToDevice);
    }

    // Now copy the lightsConfig pointer from host to device sceneConfig
    hipMemcpy(&d_sceneCfg->lightsCfg, d_lightsCfg, sizeof(lightsConfig), hipMemcpyHostToDevice);




    // Building the world
    hittable_list **elist;
    checkCudaErrors(hipMalloc((void**)&elist, sizeof(hittable_list*)));

    hittable_list **elights;
    checkCudaErrors(hipMalloc((void**)&elights, sizeof(hittable_list*)));
    
    camera** cam;
    checkCudaErrors(hipMalloc((void**)&cam, sizeof(camera*)));

    sampler** aa_sampler;
    checkCudaErrors(hipMalloc((void**)&aa_sampler, sizeof(sampler*)));


    //scene** myscene;
    //checkCudaErrors(hipMalloc((void**)&myscene, sizeof(scene*)));


    load_scene<<<single_block, single_thread>>>(d_sceneCfg, elist, elights, cam, aa_sampler, width, height, ratio, spp, sqrt_spp, texture, 1984);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    dim3 render_blocks(width / tx+1, height / ty+1);
    dim3 render_threads(tx, ty);


    printf("[INFO] Render with %u/%u blocks of %u/%u threads\n", render_blocks.x, render_blocks.y, render_threads.x, render_threads.y);


    render<<<render_blocks, render_threads>>>(world_device, image, width, height, spp, sqrt_spp, max_depth, elist, elights, cam, aa_sampler, 2580);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());


    // little padding to avoid remaining black zone at the end of the render preview
    for (int jj = 0; jj < 4; jj++)
    {
        for (int ii = 0; ii < width; ii++)
        {
            printf("p %u %u %u %u %u\n", ii, height - jj - 1, 0, 0, 0);
        }
    }
    


    uint8_t* imageHost = new uint8_t[width * height * 3 * sizeof(uint8_t)];
    for (int j = height - 1; j >= 0; j--) {
        for (int i = 0; i < width; i++) {
            size_t pixel_index = j * width + i;

            imageHost[(height - j - 1) * width * 3 + i * 3] = (uint8_t)image[pixel_index].r();
            imageHost[(height - j - 1) * width * 3 + i * 3 + 1] = (uint8_t)image[pixel_index].g();
            imageHost[(height - j - 1) * width * 3 + i * 3 + 2] = (uint8_t)image[pixel_index].b();
        }
    }

    stbi_write_png(filepath, width, height, 3, imageHost, width * 3);

    // Clean up
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    
    checkCudaErrors(hipFree(cam));
    checkCudaErrors(hipFree(elights));
    checkCudaErrors(hipFree(elist));
    checkCudaErrors(hipFree(world_device));
    checkCudaErrors(hipFree(aa_sampler));
    checkCudaErrors(hipFree(image));
    checkCudaErrors(hipFree(d_sceneCfg));
}


void launchGPU(const sceneConfig& sceneCfg, int width, int height, int spp, int max_depth, int tx, int ty, const char* filepath, bool quietMode)
{
    hipDeviceProp_t prop;

    if (!isGpuAvailable(prop))
    {
        return;
    }

    //std::cout << "Rendering222 " << nx << "x" << ny << " " << ns << " samples > " << filepath << std::endl;

    //std::cout << "[INFO] Use GPU device " << deviceIndex << " " << deviceName << std::endl;

    // https://developer.nvidia.com/blog/accelerated-ray-tracing-cuda/
    // __global__ - Runs on the GPU, called from the CPU or the GPU*. Executed with <<<dim3>>> arguments.
    // __device__ - Runs on the GPU, called from the GPU. Can be used with variabiles too.
    // __host__ - Runs on the CPU, called from the CPU.
    // 
    // --expt-relaxed-constexpr -Xcudafe --diag_suppress=esa_on_defaulted_function_ignored --std c++20 --verbose
    // --expt-relaxed-constexpr --std c++20 -Xcudafe="--diag_suppress=20012 --diag_suppress=20208" 
    //
    renderGPU(sceneCfg, prop, width, height, spp, max_depth, tx, ty, filepath);
}


//int main(int argc, char* argv[])
//{
//    launchGPU(256, 144, 10, 2, 16, 16, "e:\\ttt2.png", true);
//}